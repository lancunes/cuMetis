#include "hip/hip_runtime.h"

#include"include/cuMetis.h"

int test_time;

/*Time function params*/
double part_all = 0;
struct timeval begin_part_all;
struct timeval   end_part_all;

double part_coarsen = 0;
struct timeval begin_part_coarsen;
struct timeval   end_part_coarsen;

double part_init = 0;
struct timeval begin_part_init;
struct timeval   end_part_init;

double part_uncoarsen = 0;
struct timeval begin_part_uncoarsen;
struct timeval   end_part_uncoarsen;

//four calculation pattern
double part_match = 0;
struct timeval begin_part_match;
struct timeval   end_part_match;

double part_contract = 0;
struct timeval begin_part_contract;
struct timeval   end_part_contract;
  


double part_cmatch = 0;
struct timeval begin_part_cmatch;
struct timeval   end_part_cmatch;

double part_ccontract = 0;
struct timeval begin_part_ccontract;
struct timeval   end_part_ccontract;

double part_bfs = 0;
struct timeval begin_part_bfs;
struct timeval   end_part_bfs;

double part_2refine = 0;
struct timeval begin_part_2refine;
struct timeval   end_part_2refine;

double part_2map = 0;
struct timeval begin_part_2map;
struct timeval   end_part_2map;

double part_slipt = 0;
struct timeval begin_part_slipt;
struct timeval   end_part_slipt;



double part_krefine = 0;
struct timeval begin_part_krefine;
struct timeval   end_part_krefine;

double part_map = 0;
struct timeval begin_part_map;
struct timeval   end_part_map;

//match
double scuda_match = 0;
struct timeval begin_cuda_match;
struct timeval   end_cuda_match;

double scuda_cleanv = 0;
struct timeval begin_cuda_cleanv;
struct timeval   end_cuda_cleanv;

double sfindc1 = 0;
struct timeval begin_findc1;
struct timeval   end_findc1;

double sfindc2 = 0;
struct timeval begin_findc2;
struct timeval   end_findc2;

double sinclusive_scan = 0;
struct timeval begin_inclusive_scan;
struct timeval   end_inclusive_scan;

double sfindc2_5 = 0;
struct timeval begin_findc2_5;
struct timeval   end_findc2_5;

double sfindc3 = 0;
struct timeval begin_findc3;
struct timeval   end_findc3;

double sfindc4 = 0;
struct timeval begin_findc4;
struct timeval   end_findc4;


double sexclusive_scan = 0;
struct timeval begin_exclusive_scan;
struct timeval   end_exclusive_scan;

double sfind_cnvtxsedge_original = 0;
struct timeval begin_find_cnvtxsedge_original;
struct timeval   end_find_cnvtxsedge_original;

double sbb_segsort = 0;
struct timeval begin_bb_segsort;
struct timeval   end_bb_segsort;

double sSort_cnedges_part2 = 0;
struct timeval begin_Sort_cnedges_part2;
struct timeval   end_Sort_cnedges_part2;

double sinclusive_scan2 = 0;
struct timeval begin_inclusive_scan2;
struct timeval   end_inclusive_scan2;

double sinitcuda_match = 0;
struct timeval begin_initcuda_match;
struct timeval   end_initcuda_match;

double sinitcudajs = 0;
struct timeval begin_initcudajs;
struct timeval   end_initcudajs;

double sSort_cnedges_part1 = 0;
struct timeval begin_Sort_cnedges_part1;
struct timeval   end_Sort_cnedges_part1;

double sSort_cnedges_part2_5 = 0;
struct timeval begin_Sort_cnedges_part2_5;
struct timeval   end_Sort_cnedges_part2_5;

double sSort_cnedges_part3 = 0;
struct timeval begin_Sort_cnedges_part3;
struct timeval   end_Sort_cnedges_part3;

double sCoarsen = 0;
struct timeval begin_sCoarsen;
struct timeval   end_sCoarsen;

/*Define functions*/
#define cuMetis_max(m,n) ((m)>=(n)?(m):(n))
#define cuMetis_min(m,n) ((m)>=(n)?(n):(m))
#define cuMetis_swap(m,n,temp) do{(temp)=(m);(m)=(n);(n)=(temp);} while(0) 
#define cuMetis_tocsr(i,n,c) do{for(i=1;i<n;i++)c[i]+= c[i-1];for(i=n;i>0;i--)c[i]=c[i-1];c[0]=0;} while(0) 
#define cuMetis_add_sub(m,n,temp) do{(m)+=(temp);(n)-=(temp);} while(0)
#define cuMetis_listinsert(n,list,lptr,i) do{list[n]=i;lptr[i]=(n)++;} while(0) 
#define cuMetis_listdelete(n,list,lptr,i) do{list[lptr[i]]=list[--(n)];lptr[list[n]]=lptr[i];lptr[i]=-1;} while(0) 
#define M_GT_N(m,n) ((m)>(n))


/*Graph data structure*/
typedef struct cuMetis_graph_t {
  /*graph cpu params*/
  int nvtxs;                            //Graph vertex
  int nedges;	                          //Graph edge
  int *xadj;                            //Graph vertex csr array (xadj[nvtxs+1])
  int *adjncy;                          //Graph adjacency list (adjncy[nedges])
  int *adjwgt;   		                    //Graph edge weight array (adjwgt[nedges])
  int *vwgt;			                      //Graph vertex weight array(vwgr[nvtxs])
  int *tvwgt;                           //The sum of graph vertex weight 
  float *tvwgt_reverse;                 //The reciprocal of tvwgt
  int *label;                           //Graph vertex label(label[nvtxs])
  int *cmap;                            //The Label of graph vertex in cgraph(cmap[nvtxs]) 
  int mincut;                           //The min edfe-cut of graph partition
  int *where;                           //The label of graph vertex in which part(where[nvtxs]) 
  int *pwgts;                           //The partition vertex weight(pwgts[nparts])
  int nbnd;                             //Boundary vertex number
  int *bndlist;                         //Boundary vertex list
  int *bndptr;                          //Boundary vertex pointer
  int *id;                              //The sum of edge weight in same part
  int *ed;                              //The sum of edge weight in different part
  struct cuMetis_graph_t *coarser; //The coarser graph
  struct cuMetis_graph_t *finer;   //The finer graph
  /*graph gpu params*/
  int *cuda_nvtxs;
  int *cuda_xadj;
  int *cuda_adjncy;
  int *cuda_adjwgt;
  int *cuda_vwgt;               
  int *cuda_match;                      //CUDA graph vertex match array(match[nvtxs])
  int *cuda_cmap;
  int *cuda_maxvwgt;                    //CUDA graph constraint vertex weight 
  int *cuda_real_edge;                  //CUDA graph vertex pairs csr edge array(cuda_real_edge[cnvtxs+1])
  int *cuda_real_nvtxs;                 //CUDA graph params (i<match[cmap[i]])
  int *cuda_cnvtxs;                     //CUDA coarsen graph vertex
  int *cuda_s;                          //CUDA support array (cuda_s[nvtxs])
  int *cuda_scan_adjwgt_original;       //CUDA support scan array (cuda_scan_adjwgt_original[nedges])
  int *cuda_scan_nedges_original;       //CUDA support scan array (cuda_scan_nedges_original[nedges])
  int *cuda_js;                         //CUDA support array (cuda_js[cnvtxs])
  int *cuda_scan_cnedges_original;      //CUDA support scan array (cuda_scan_cnedges_original[nedges])
  int *cuda_maxwgt;                     //CUDA part weight array (cuda_maxwgt[npart])
  int *cuda_minwgt;                     //CUDA part weight array (cuda_minwgt[npart])
  int *cuda_where;
  int *cuda_pwgts;
  int *cuda_bnd;
  int *cuda_bndnum;
  int *cpu_bndnum;
  int *cuda_info;                       //CUDA support array(cuda_info[bnd_num*nparts])
  int *cuda_real_bnd_num;
  int *cuda_real_bnd;
  int *cuda_nparts;
  int *cuda_tvwgt;
  float *cuda_tpwgts;
} cuMetis_graph_t;

/*Refinement available generate array*/
int *cu_bn;                             
int *cu_bt;
int *cu_g;
int *cu_csr;
int *cu_que;

/*Memory allocation information*/
typedef struct cuMetis_mop_t {
  int type;
  ssize_t nbytes;
  void *ptr;
} cuMetis_mop_t;

/*Algorithm information*/
typedef struct cuMetis_mcore_t {
  void *core;	
  size_t coresize;     
  size_t corecpos;            
  size_t nmops;         
  size_t cmop;         
  cuMetis_mop_t *mops;      
  size_t num_callocs;   
  size_t num_hallocs;   
  size_t size_callocs;  
  size_t size_hallocs;  
  size_t cur_callocs;   
  size_t cur_hallocs;  
  size_t max_callocs;   
  size_t max_hallocs;   

} cuMetis_mcore_t;

/*Control information*/
typedef struct cuMetis_admin_t {
  int Coarsen_threshold;		
  int nIparts;                                                                                                                                       
  int iteration_num;                               
  int *maxvwgt;		                
  int nparts;                	
  float *ubfactors;            
  float *tpwgts;               
  float *part_balance;               
  float cfactor;               
  cuMetis_mcore_t *mcore;    
  size_t nbrpoolsize;      
  size_t nbrpoolcpos;                  

} cuMetis_admin_t;


/*Heap information*/
typedef struct cuMetis_rkv_t{
  float key;
  int val;
} cuMetis_rkv_t;


/*Queue information*/
typedef struct {
  ssize_t nnodes;
  ssize_t maxnodes;
  cuMetis_rkv_t   *heap;
  ssize_t *locator;
} cuMetis_queue_t;

/*Compute log2 algorithm*/
int cuMetis_compute_log2(int a)
{
  int i;
  for(i=1;a>1;i++,a=a>>1);
  return i-1;
}

/*Get int rand number*/
int cuMetis_int_rand() 
{
  if(sizeof(int)<=sizeof(int32_t)) 
    return (int)(uint32_t)rand();
  else  
    return (int)(uint64_t)rand(); 
}


/*Get int rand number between (0,max)*/
int cuMetis_int_randinrange(int max) 
{
  return (int)((cuMetis_int_rand())%max); 
}


/*Compute sum of int array*/
int cuMetis_int_sum(size_t n, int *a)
{
  size_t i;
  int sum=0;
  for(i=0;i<n;i++,a+=1){
    sum+=(*a);
  }
  return sum;
}

/*Copy int array a to b*/
int  *cuMetis_int_copy(size_t n, int *a, int *b)
{
  return (int *)memmove((void *)b, (void *)a, sizeof(int)*n);
}


/*Set int array value*/
int *cuMetis_int_set_value(size_t n, int val, int *a)
{
  size_t i;
  for(i=0;i<n;i++){
    a[i]=val;
  }
  return a;
}


/*Compute sum of float array*/
float cuMetis_float_sum(size_t n, float *a)
{
  size_t i;
  float sum=0;
  for(i=0;i<n;i++,a+=1){
    sum+=(*a);
  }
  return sum;
}


/*Rescale tpwgts array*/
float *cuMetis_tpwgts_rescale(size_t n, float wsum, float *a)
{
  size_t i;
  for(i=0;i<n;i++,a+=1){
    (*a)*=wsum;
  }
  return a;
}


/*Compute Partition result edge-cut*/
int cuMetis_computecut(cuMetis_graph_t *graph, int *where)
{
  int i,j,cut=0;
    for(i=0;i<graph->nvtxs;i++){
      for(j=graph->xadj[i];j<graph->xadj[i+1];j++)
        if(where[i]!=where[graph->adjncy[j]]){
          cut+=graph->adjwgt[j];
        }
    }
  return cut/2;
}


/*Set graph admin params*/
cuMetis_admin_t *cuMetis_set_graph_admin(int nparts, float *tpwgts, float *ubvec)
{
  int i;
  cuMetis_admin_t *cuMetis_admin;
  cuMetis_admin=(cuMetis_admin_t *)malloc(sizeof(cuMetis_admin_t));
  memset((void *)cuMetis_admin,0,sizeof(cuMetis_admin_t));

  cuMetis_admin->iteration_num=10;
  cuMetis_admin->Coarsen_threshold=200;
  cuMetis_admin->nparts=nparts; 

  cuMetis_admin->maxvwgt=(int*)malloc(sizeof(int));
  cuMetis_admin->maxvwgt[0]=0;  

  cuMetis_admin->tpwgts=(float*)malloc(sizeof(float)*nparts);
  for(i=0;i<nparts;i++){
    cuMetis_admin->tpwgts[i]=1.0/nparts;
  }

  cuMetis_admin->ubfactors=(float*)malloc(sizeof(float));
  cuMetis_admin->ubfactors[0] =1.03;

  cuMetis_admin->part_balance =(float*) malloc(sizeof(float)*nparts);
  return cuMetis_admin;  
}


/*Set graph params*/
void cuMetis_init_cpu_graph(cuMetis_graph_t *graph) 
{
  memset((void *)graph,0,sizeof(cuMetis_graph_t));
  graph->nvtxs     = -1;
  graph->nedges    = -1;
  graph->xadj      = NULL;
  graph->vwgt      = NULL;
  graph->adjncy    = NULL;
  graph->adjwgt    = NULL;
  graph->label     = NULL;
  graph->cmap      = NULL;
  graph->tvwgt     = NULL;
  graph->tvwgt_reverse  = NULL;
  graph->where     = NULL;
  graph->pwgts     = NULL;
  graph->mincut    = -1;
  graph->nbnd      = -1;
  graph->id        = NULL;
  graph->ed        = NULL;
  graph->bndptr    = NULL;
  graph->bndlist   = NULL;
  graph->coarser   = NULL;
  graph->finer     = NULL;
}


/*Malloc graph*/
cuMetis_graph_t *cuMetis_create_cpu_graph(void)
{
  cuMetis_graph_t *graph;
  graph=(cuMetis_graph_t *)malloc(sizeof(cuMetis_graph_t));
  cuMetis_init_cpu_graph(graph);
  return graph;
}


/*Set graph tvwgt value*/
void cuMetis_set_graph_tvwgt(cuMetis_graph_t *graph)
{
  if(graph->tvwgt==NULL){ 
    graph->tvwgt=(int*)malloc(sizeof(int));
  }

  if(graph->tvwgt_reverse==NULL){ 
    graph->tvwgt_reverse=(float*)malloc(sizeof(float));
  }

  graph->tvwgt[0]=cuMetis_int_sum(graph->nvtxs,graph->vwgt);
  graph->tvwgt_reverse[0]=1.0/(graph->tvwgt[0]>0?graph->tvwgt[0]:1);
}


/*Set graph vertex label*/
void cuMetis_set_graph_label(cuMetis_graph_t *graph)
{
  int i;

  if(graph->label==NULL){
    graph->label=(int*)malloc(sizeof(int)*(graph->nvtxs));
  }

  for(i=0;i<graph->nvtxs;i++){
    graph->label[i]=i;
  }

}


/*Set graph information*/
cuMetis_graph_t *cuMetis_set_graph(cuMetis_admin_t *cuMetis_admin, int nvtxs, \
int *xadj, int *adjncy, int *vwgt , int *adjwgt) 
{
  int i;
  cuMetis_graph_t *graph;
  
  graph = cuMetis_create_cpu_graph();
  graph->nvtxs=nvtxs;
  graph->nedges=xadj[nvtxs];
  graph->xadj=xadj;
  graph->adjncy=adjncy;
  
  if(vwgt){
    graph->vwgt=vwgt;
  }
  else{
    vwgt=graph->vwgt=(int*)malloc(sizeof(int)*nvtxs);
    for(i=0;i<nvtxs;i++){
      vwgt[i]=graph->vwgt[i]=1;
    }
  
  }
  
  graph->tvwgt=(int*)malloc(sizeof(int));
  graph->tvwgt_reverse=(float*)malloc(sizeof(float));
  graph->tvwgt[0]=cuMetis_int_sum(nvtxs, vwgt);
  graph->tvwgt_reverse[0]=1.0/(graph->tvwgt[0]>0?graph->tvwgt[0]:1);

  if(adjwgt){
    graph->adjwgt=adjwgt;
  }
  else{
    adjwgt=graph->adjwgt=(int*)malloc(sizeof(int)*(graph->nedges));
    for(i=0;i<graph->nedges;i++){
      adjwgt[i]=graph->adjwgt[i]=1;
    }
  }
  
  cuMetis_set_graph_tvwgt(graph);
  cuMetis_set_graph_label(graph);
  
  return graph;
}


/*Creates mcore*/
cuMetis_mcore_t *cuMetis_create_mcore(size_t coresize)
{
  cuMetis_mcore_t *mcore;
  mcore=(cuMetis_mcore_t *)malloc(sizeof(cuMetis_mcore_t));
  memset(mcore,0,sizeof(cuMetis_mcore_t));

  mcore->coresize=coresize;
  mcore->corecpos=0;
  mcore->core=(coresize==0?NULL:(size_t*)malloc(sizeof(size_t)*(mcore->coresize)));
  mcore->nmops=2048;
  mcore->cmop=0;
  mcore->mops=(cuMetis_mop_t *)malloc((mcore->nmops)*sizeof(cuMetis_mop_t));

  return mcore;
}


/*Allocate work space*/
void cuMetis_allocatespace(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  size_t coresize;
  coresize=3*(graph->nvtxs+1)*sizeof(int)+5*(cuMetis_admin->nparts+1)*sizeof(int)\
  +5*(cuMetis_admin->nparts+1)*sizeof(float);

  cuMetis_admin->mcore=cuMetis_create_mcore(coresize);
  cuMetis_admin->nbrpoolsize=0;
  cuMetis_admin->nbrpoolcpos=0;
}


/*Add memory allocation*/
void cuMetis_add_mcore(cuMetis_mcore_t *mcore, int type, size_t nbytes, void *ptr)
{
  if(mcore->cmop==mcore->nmops){
    mcore->nmops*=2;
    mcore->mops=(cuMetis_mop_t*)realloc(mcore->mops, mcore->nmops*sizeof(cuMetis_mop_t));
    if(mcore->mops==NULL){
      exit(0);
    }
  }

  mcore->mops[mcore->cmop].type=type;
  mcore->mops[mcore->cmop].nbytes=nbytes;
  mcore->mops[mcore->cmop].ptr=ptr;
  mcore->cmop++;

  switch(type){
    case 1:
      break;
    
    case 2:
      mcore->num_callocs++;
      mcore->size_callocs+=nbytes;
      mcore->cur_callocs+=nbytes;
      if(mcore->max_callocs<mcore->cur_callocs){
        mcore->max_callocs=mcore->cur_callocs;
      }
      break;
    
    case 3:
      mcore->num_hallocs++;
      mcore->size_hallocs+=nbytes;
      mcore->cur_hallocs+=nbytes;
      if(mcore->max_hallocs<mcore->cur_hallocs){
        mcore->max_hallocs=mcore->cur_hallocs;
      }
      break;
    
    default:
      exit(0);
  }
}


/*Malloc mcore*/
void *cuMetis_malloc_mcore(cuMetis_mcore_t *mcore, size_t nbytes)
{
  void *ptr;
  nbytes+=(nbytes%8==0?0:8-nbytes%8);

  if(mcore->corecpos+nbytes<mcore->coresize){
    ptr=((char *)mcore->core)+mcore->corecpos;
    mcore->corecpos+=nbytes;
    cuMetis_add_mcore(mcore,2,nbytes,ptr);
  }
  else{
    ptr=(size_t*)malloc(nbytes);
    cuMetis_add_mcore(mcore,3,nbytes,ptr);
  }

  return ptr;
}


/*Malloc mcore space*/
void *cuMetis_malloc_space(cuMetis_admin_t *cuMetis_admin, size_t nbytes)
{
  return cuMetis_malloc_mcore(cuMetis_admin->mcore,nbytes);
}


/*Malloc int mcore space*/
int *cuMetis_int_malloc_space(cuMetis_admin_t *cuMetis_admin, size_t n)
{
  return (int *)cuMetis_malloc_space(cuMetis_admin, n*sizeof(int));
}


/*Malloc float mcore space*/
float *cuMetis_float_malloc_space(cuMetis_admin_t *cuMetis_admin)
{
  return (float *)cuMetis_malloc_space(cuMetis_admin,2*sizeof(float));
}


/*Compute 2way balance params*/
void cuMetis_compute_2way_balance(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph, float *tpwgts)
{
  int i;
  for(i=0;i<2;i++){
      cuMetis_admin->part_balance[i]=graph->tvwgt_reverse[0]/tpwgts[i];
  }
}


/*Get random permute of p*/
void cuMetis_int_randarrayofp(int n, int *p, int m, int flag)
{
  int i,u,v;
  int temp;
  if(flag==1){
    for(i=0;i<n;i++)
      p[i] = (int)i;
  }

  if(n<10){
    for(i=0;i<n;i++){

      v=cuMetis_int_randinrange(n);
      u=cuMetis_int_randinrange(n);
     
      cuMetis_swap(p[v],p[u],temp);

    }
  }
  else{
    for(i=0;i<m;i++){

      v=cuMetis_int_randinrange(n-3);
      u=cuMetis_int_randinrange(n-3);
      
      cuMetis_swap(p[v+0],p[u+2],temp);
      cuMetis_swap(p[v+1],p[u+3],temp);
      cuMetis_swap(p[v+2],p[u+0],temp);
      cuMetis_swap(p[v+3],p[u+1],temp);

    }
  }
}


/*Get permutation array*/
void cuMetis_matching_sort(cuMetis_admin_t *cuMetis_admin, int n, \
int max, int *keys, int *tperm, int *perm)
{
  int i,ii;
  int *counts;
  counts=cuMetis_int_set_value(max+2,0,cuMetis_int_malloc_space(cuMetis_admin,max+2));
  
  for(i=0; i<n; i++){
    counts[keys[i]]++;
  }
  
  cuMetis_tocsr(i,max+1,counts);
  
  for(ii=0;ii<n;ii++){
    i=tperm[ii];
    perm[counts[keys[i]]++]=i;
  }
}


/*Malloc cpu coarsen graph params*/
cuMetis_graph_t *cuMetis_set_cpu_cgraph(cuMetis_graph_t *graph, int cnvtxs)
{
  cuMetis_graph_t *cgraph;
  cgraph=cuMetis_create_cpu_graph();
  
  cgraph->nvtxs=cnvtxs;
  cgraph->xadj=(int*)malloc(sizeof(int)*(cnvtxs+1));
  cgraph->adjncy=(int*)malloc(sizeof(int)*(graph->nedges));
  cgraph->adjwgt=(int*)malloc(sizeof(int)*(graph->nedges));
  cgraph->vwgt=(int*)malloc(sizeof(int)*cnvtxs);
  cgraph->tvwgt=(int*)malloc(sizeof(int));
  cgraph->tvwgt_reverse=(float*)malloc(sizeof(float)); 
  
  cgraph->finer=graph;
  graph->coarser=cgraph;
  
  return cgraph;
}



/*Malloc gpu coarsen graph params*/
cuMetis_graph_t *cuMetis_set_gpu_cgraph(cuMetis_graph_t *graph, int cnvtxs)
{
  cuMetis_graph_t *cgraph;
  cgraph=cuMetis_create_cpu_graph();
  
  cgraph->nvtxs=cnvtxs;
  cgraph->xadj=(int*)malloc(sizeof(int)*(cnvtxs+1));
  cgraph->tvwgt=(int*)malloc(sizeof(int));
  cgraph->tvwgt_reverse=(float*)malloc(sizeof(float)); 
  
  cgraph->finer=graph;
  graph->coarser=cgraph;
  
  return cgraph;
}



/*Create cpu coarsen graph by contract*/
void cuMetis_cpu_create_cgraph(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, int cnvtxs, int *match,int level)
{
  int j,k,m,istart,iend,nvtxs,nedges,cnedges,v,u;
  int *xadj,*vwgt,*adjncy,*adjwgt;
  int *cmap,*htable;
  int *cxadj,*cvwgt,*cadjncy,*cadjwgt;
  cuMetis_graph_t *cgraph;
  
  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  cmap=graph->cmap;                  
  
  cgraph=cuMetis_set_cpu_cgraph(graph,cnvtxs);            
  cxadj=cgraph->xadj;
  cvwgt=cgraph->vwgt;
  cadjncy=cgraph->adjncy;
  cadjwgt=cgraph->adjwgt;                               
  htable=cuMetis_int_set_value(cnvtxs,-1,cuMetis_int_malloc_space(cuMetis_admin,cnvtxs));      
  cxadj[0] = cnvtxs = cnedges = 0; 
  nedges=graph->nedges;
   
  for(v=0;v<nvtxs;v++){

    if((u=match[v])<v)         
      continue;   

    cvwgt[cnvtxs]=vwgt[v];                 
    nedges=0;                                                    
    istart=xadj[v];
    iend=xadj[v+1];    

    for(j=istart;j<iend;j++){

      k=cmap[adjncy[j]];     

      if((m=htable[k])==-1){
        cadjncy[nedges]=k;                           
        cadjwgt[nedges] = adjwgt[j];                      
        htable[k] = nedges++;  
      }
      else{
        cadjwgt[m] += adjwgt[j];                                 
      }
    }

    if(v!=u){ 
      cvwgt[cnvtxs]+=vwgt[u];                   
      istart=xadj[u];                                    
      iend=xadj[u+1];      

      for(j=istart;j<iend;j++){
        k=cmap[adjncy[j]];

        if((m=htable[k])==-1){
          cadjncy[nedges]=k;
          cadjwgt[nedges]=adjwgt[j];
          htable[k]=nedges++;
        }
        else{
          cadjwgt[m] += adjwgt[j];
        }
      }

      if((j=htable[cnvtxs])!=-1){
        cadjncy[j]=cadjncy[--nedges];
        cadjwgt[j]=cadjwgt[nedges];
        htable[cnvtxs] = -1;
      }
    }

    for(j=0;j<nedges;j++){
       htable[cadjncy[j]] = -1;  
    }

    cnedges+=nedges;
    cxadj[++cnvtxs]=cnedges;
    cadjncy+=nedges;                                                                 
    cadjwgt+=nedges;
  }

  cgraph->nedges=cnedges;
  cgraph->tvwgt[0]=cuMetis_int_sum(cgraph->nvtxs,cgraph->vwgt); 
  cgraph->tvwgt_reverse[0]=1.0/(cgraph->tvwgt[0]>0?cgraph->tvwgt[0]:1);    

}



/*Get cpu graph matching params by hem*/
int cuMetis_cpu_match(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph,int level)
{
  hipDeviceSynchronize();
  gettimeofday(&begin_part_cmatch,NULL);

  int i,j,pi,k,nvtxs,cnvtxs,maxidx,maxwgt,aved;
  int *xadj,*vwgt,*adjncy,*adjwgt,*maxvwgt;
  int *match,*cmap,*d,*perm,*tperm;

  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  cmap=graph->cmap;
  maxvwgt=cuMetis_admin->maxvwgt;
  
  cnvtxs=0;
  match=cuMetis_int_set_value(nvtxs,-1, cuMetis_int_malloc_space(cuMetis_admin,nvtxs));
  perm=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  tperm=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  d=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);         
  cuMetis_int_randarrayofp(nvtxs,tperm,nvtxs/8,1);   
  aved=0.7*(xadj[nvtxs]/nvtxs);

  for(i=0;i<nvtxs;i++){ 
    d[i]=(xadj[i+1]-xadj[i]>aved?aved:xadj[i+1]-xadj[i]);
  }

  cuMetis_matching_sort(cuMetis_admin,nvtxs,aved,d,tperm,perm);         
  
  for(pi=0;pi<nvtxs;pi++) 
  {
    i=perm[pi];  

    if(match[i]==-1){  
      maxidx=i;                                                                               
      maxwgt=-1;           

      for(j=xadj[i];j<xadj[i+1];j++){
        k=adjncy[j];

        if(match[k]==-1&&maxwgt<adjwgt[j]&&vwgt[i]+vwgt[k]<=maxvwgt[0]){
          maxidx=k;
          maxwgt=adjwgt[j];
        }   

        if(maxidx==i&&3*vwgt[i]<maxvwgt[0]){ 
          maxidx=-1;
        }
      } 

      if(maxidx!=-1){
        cmap[i]=cmap[maxidx]=cnvtxs++;              
        match[i]=maxidx;                                        
        match[maxidx]=i; 
      }
    }
 
  }         

  for(cnvtxs=0,i=0;i<nvtxs;i++){
    if(match[i]==-1){
      match[i]=i;
      cmap[i]=cnvtxs++;                                                    
    }
    else{
      if(i<=match[i]){ 
        cmap[i]=cmap[match[i]]=cnvtxs++;
      }
    }
  }

  hipDeviceSynchronize();
  gettimeofday(&end_part_cmatch,NULL);
  part_cmatch += (end_part_cmatch.tv_sec - begin_part_cmatch.tv_sec) * 1000 + (end_part_cmatch.tv_usec - begin_part_cmatch.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_part_ccontract,NULL);

  cuMetis_cpu_create_cgraph(cuMetis_admin, graph, cnvtxs, match,level);
  
  hipDeviceSynchronize();
  gettimeofday(&end_part_ccontract,NULL);
  part_ccontract += (end_part_ccontract.tv_sec - begin_part_ccontract.tv_sec) * 1000 + (end_part_ccontract.tv_usec - begin_part_ccontract.tv_usec) / 1000.0;

  return cnvtxs;
}



/*Malloc and memcpy original graph from cpu to gpu*/
void cuMetis_malloc_original_coarseninfo(cuMetis_admin_t *cuMetis_admin,cuMetis_graph_t *graph)
{
    int nvtxs=graph->nvtxs;
    int nedges=graph->nedges;

    hipMalloc((void**)&graph->cuda_nvtxs,sizeof(int));
    hipMemcpy(graph->cuda_nvtxs,&graph->nvtxs,sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_match,nvtxs*sizeof(int));

    hipMalloc((void**)&graph->cuda_xadj,(nvtxs+1)*sizeof(int));
    hipMemcpy(graph->cuda_xadj,graph->xadj,(nvtxs+1)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_vwgt,(nvtxs+1)*sizeof(int));
    hipMemcpy(graph->cuda_vwgt,graph->vwgt,nvtxs*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_adjncy,nedges*sizeof(int));
    hipMemcpy(graph->cuda_adjncy,graph->adjncy,nedges*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_adjwgt,nedges*sizeof(int));
    hipMemcpy(graph->cuda_adjwgt,graph->adjwgt,nedges*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_cmap,nvtxs*sizeof(int));

    hipMalloc((void**)&graph->cuda_maxvwgt,sizeof(int));

    hipMalloc((void**)&graph->cuda_s,(nvtxs)*sizeof(int));

    hipMalloc((void**)&graph->cuda_cnvtxs,sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_nedges_original,(graph->nedges)*sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_cnedges_original,(graph->nedges)*sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_adjwgt_original,(graph->nedges)*sizeof(int));

}



/*Malloc gpu coarsen graph params*/
  void cuMetis_malloc_coarseninfo(cuMetis_admin_t *cuMetis_admin,cuMetis_graph_t *graph)
{
    int nvtxs=graph->nvtxs;
    int nedges=graph->nedges;

    hipMalloc((void**)&graph->cuda_nvtxs,sizeof(int));
    hipMemcpy(graph->cuda_nvtxs,&graph->nvtxs,sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&graph->cuda_match,nvtxs*sizeof(int));

    hipMalloc((void**)&graph->cuda_cmap,nvtxs*sizeof(int));

    hipMalloc((void**)&graph->cuda_maxvwgt,sizeof(int)); 

    hipMalloc((void**)&graph->cuda_s,(nvtxs)*sizeof(int));

    hipMalloc((void**)&graph->cuda_cnvtxs,sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_nedges_original,nedges*sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_cnedges_original,nedges*sizeof(int));

    hipMalloc((void**)&graph->cuda_scan_adjwgt_original,nedges*sizeof(int));

}


/*CUDA-initial cuda_js array*/
__global__ void initcudajs(int *cuda_js, int *cn)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<cn[0]){
  cuda_js[ii]=0;
  }

}



/*CUDA-set each vertex pair adjacency list and weight params*/
__global__ void find_cnvtxsedge_original(int *cuda_scan_nedges, int *cuda_scan_order, int *cuda_xadj,\
  int *cuda_match, int *cuda_adjncy, int *cuda_scan_nedges_original, int *cuda_cmap, int *cvwgt, int *vwgt, int *js,\
  int *cuda_scan_cnedges_original, int *cuda_scan_adjwgt_original, int *cuda_adjwgt)
{
  int pi,u,istart,iend,i,pp;
  pi=blockIdx.x;
  int tid=threadIdx.x;
  u=cuda_match[pi];  

  if(pi>u){
    pp=cuda_scan_nedges[cuda_cmap[pi]]+cuda_xadj[u+1]-cuda_xadj[u];
  }
  else{
    pp=cuda_scan_nedges[cuda_cmap[pi]];
  }

  int sum=(cuda_xadj[pi+1]-cuda_xadj[pi]);
  if(sum<32){

    if(threadIdx.x<sum){
      istart=cuda_xadj[pi]+threadIdx.x;
      iend=cuda_xadj[pi]+threadIdx.x+1;

      for(i=istart;i<iend;i++){
        int pt=pp+i-cuda_xadj[pi];
        cuda_scan_nedges_original[pt]=cuda_adjncy[i];
        cuda_scan_cnedges_original[pt]=cuda_cmap[cuda_adjncy[i]];
        cuda_scan_adjwgt_original[pt]=cuda_adjwgt[i];
      } 
    }

  }
  else{
    int tt=32;
    int b=sum/tt;
    int a=b+1;
    int x=sum-b*tt;

    if(threadIdx.x<x){
      istart=cuda_xadj[pi]+threadIdx.x*a;
      iend=cuda_xadj[pi]+(threadIdx.x+1)*a;
    }
    else{
      istart=cuda_xadj[pi]+x*a+(threadIdx.x-x)*b;
      iend=cuda_xadj[pi]+x*a+(threadIdx.x+1-x)*b;
    }
    for(i=istart;i<iend;i++){
      int pt=pp+i-cuda_xadj[pi];

      cuda_scan_nedges_original[pt]=cuda_adjncy[i];
      cuda_scan_cnedges_original[pt]=cuda_cmap[cuda_adjncy[i]];
      cuda_scan_adjwgt_original[pt]=cuda_adjwgt[i];
    } 
  } 
  if(tid==0){
    if(u!=pi){
      cvwgt[cuda_cmap[pi]]=vwgt[pi]+vwgt[u];
    }
    else{
      cvwgt[cuda_cmap[pi]]=vwgt[pi];
    }
  }
}


/*CUDA-Segmentation sorting part1-set scan array value 0 or 1*/
__global__ void Sort_cnedges2_part1(int *cuda_scan_cnedges_original,\
int *cuda_scan_nedges, int *cuda_scan_order, int *cuda_cmap, int *temp_scan)
{
  int pi,istart,iend,i;
  pi=blockIdx.x;
  int tid=threadIdx.x; 
  int pp,ppp;
  int pii;

  pii=cuda_scan_order[pi];
  pp=cuda_scan_nedges[cuda_cmap[pii]];
  ppp=cuda_scan_nedges[cuda_cmap[pii]+1];

  int sum=ppp-pp;

  if(sum<32){
    if(threadIdx.x<sum){
      istart=tid;
      iend=tid+1;

      for(i=istart;i<iend;i++){
        if(i==0){
          if(cuda_scan_cnedges_original[pp+i]==cuda_cmap[pii]){
            temp_scan[pp+i]=0;
          }
          else{
            temp_scan[pp+i]=1;
          }
        }
        else{
          if(cuda_scan_cnedges_original[pp+i]==cuda_cmap[pii]){
            temp_scan[pp+i]=0;
          }
          else{
            if(cuda_scan_cnedges_original[pp+i]==cuda_scan_cnedges_original[pp+i-1]){
              temp_scan[pp+i]=0;
            }
            else{
              temp_scan[pp+i]=1;
            }
          }
        }
      }
    }
  }
  else{
    int tt=32;
    int b=sum/tt;
    int a=b+1;
    int x=sum-b*tt;

    if(threadIdx.x<x){
      istart=threadIdx.x*a;
      iend=(threadIdx.x+1)*a;
    }
    else{
      istart=x*a+(threadIdx.x-x)*b;
      iend=x*a+(threadIdx.x+1-x)*b;
    }
    
    for(i=istart;i<iend;i++){
      if(i==0){
        if(cuda_scan_cnedges_original[pp+i]==cuda_cmap[pii]){
          temp_scan[pp+i]=0;
        }
        else{
          temp_scan[pp+i]=1;
        }
      }
      else{
        if(cuda_scan_cnedges_original[pp+i]==cuda_cmap[pii]){
          temp_scan[pp+i]=0;
        }
        else{
          if(cuda_scan_cnedges_original[pp+i]==cuda_scan_cnedges_original[pp+i-1]){
            temp_scan[pp+i]=0;
          }
          else{
            temp_scan[pp+i]=1;
          }
        }
      }
    }
  }
}


/*CUDA-Segmentation sorting part2-set cxadj*/
__global__ void Sort_cnedges2_part2(int *cuda_scan_nedges, int *cuda_scan_order, int *cuda_cmap, \
int *temp_scan, int *temp_xadj, int *cn)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<cn[0]){ 
    int ppp;
    int pii;  
    pii=cuda_scan_order[pi];
    ppp=cuda_scan_nedges[cuda_cmap[pii]+1];
      
  if(pi==0){
    temp_xadj[0]=0;
  }

  temp_xadj[pi+1]=temp_scan[ppp-1];

  }
} 


/*CUDA-Segmentation sorting part2.5-init cadjwgt and cadjncy*/
__global__ void Sort_cnedges2_part2_5(int *cadjwgt, int *cadjncy, int *c)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;  

  if(pi<c[0]){ 
    cadjwgt[pi]=0;
    cadjncy[pi]=-1;
  }
}


/*CUDA-Segmentation sorting part3-deduplication and accumulation*/
__global__ void Sort_cnedges2_part3(int *cuda_scan_cnedges_original,int *cuda_scan_nedges, \
int *cuda_scan_order, int *cuda_cmap,  int *cuda_scan_adjwgt_original,int *temp_scan, int *cadjncy, int *cadjwgt)
{
  int pi,istart,iend,i;
  pi=blockIdx.x;
  int tid=threadIdx.x;
  int pp,ppp;
  int pii;

  pii=cuda_scan_order[pi];
  pp=cuda_scan_nedges[cuda_cmap[pii]];
  ppp=cuda_scan_nedges[cuda_cmap[pii]+1];

  int sum=ppp-pp;

  if(sum<32){
    if(threadIdx.x<sum){
      istart=tid;
      iend=tid+1;

      for(i=istart;i<iend;i++){
        if(i==0){
          if(cuda_scan_cnedges_original[pp+i]!=cuda_cmap[pii]){
            cadjncy[temp_scan[pp+i]-1]=cuda_scan_cnedges_original[pp+i];
            atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
          }
        }
        else{
          if(cuda_scan_cnedges_original[pp+i]!=cuda_cmap[pii]){
            if(cuda_scan_cnedges_original[pp+i]!=cuda_scan_cnedges_original[pp+i-1]){
              cadjncy[temp_scan[pp+i]-1]=cuda_scan_cnedges_original[pp+i];
              atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
            }
            else{
              atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
            }
          }
        }   
      }
    }
  }
  else {
    int tt=32;
    int b=sum/tt;
    int a=b+1;
    int x=sum-b*tt;

    if(threadIdx.x<x){
      istart=threadIdx.x*a;
      iend=(threadIdx.x+1)*a;
    }
    else{
      istart=x*a+(threadIdx.x-x)*b;
      iend=x*a+(threadIdx.x+1-x)*b;
    }
    for(i=istart;i<iend;i++){
      if(i==0){
        if(cuda_scan_cnedges_original[pp+i]!=cuda_cmap[pii]){
          cadjncy[temp_scan[pp+i]-1]=cuda_scan_cnedges_original[pp+i];
          atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
        }
      }
      else{
        if(cuda_scan_cnedges_original[pp+i]!=cuda_cmap[pii]){
          if(cuda_scan_cnedges_original[pp+i]!=cuda_scan_cnedges_original[pp+i-1]){
            cadjncy[temp_scan[pp+i]-1]=cuda_scan_cnedges_original[pp+i];
            atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
          }
          else{
            atomicAdd(&cadjwgt[temp_scan[pp+i]-1],cuda_scan_adjwgt_original[pp+i]);
          }
        }
      }     
    }
  }
}


/*Free cuda coarsen graph params*/
void cuMetis_free_coarsen(cuMetis_graph_t *graph)
{
  hipFree(graph->cuda_maxvwgt);
  hipFree(graph->cuda_match);
  hipFree(graph->cuda_real_edge);
  hipFree(graph->cuda_real_nvtxs);
  hipFree(graph->cuda_s);
  hipFree(graph->cuda_scan_adjwgt_original);
  hipFree(graph->cuda_scan_nedges_original);
  hipFree(graph->cuda_js);
  hipFree(graph->cuda_scan_cnedges_original);
}



/*Create gpu coarsen graph by contract*/
void cuMetis_gpu_create_cgraph(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, int cnvtxs, int level,int *scan_edge)
{
  int nvtxs=graph->nvtxs;
  int nedges=graph->nedges;

  cuMetis_graph_t *cgraph;
  cgraph = cuMetis_set_gpu_cgraph(graph, cnvtxs); 

  int length=cnvtxs+1; 

  hipDeviceSynchronize();
  gettimeofday(&begin_exclusive_scan,NULL);
  thrust::exclusive_scan(scan_edge,scan_edge+length,scan_edge);
  hipDeviceSynchronize();//计算临界边 索引
  gettimeofday(&end_exclusive_scan,NULL);
  sexclusive_scan += (end_exclusive_scan.tv_sec - begin_exclusive_scan.tv_sec) * 1000 + (end_exclusive_scan.tv_usec - begin_exclusive_scan.tv_usec) / 1000.0;

  hipMemcpy( graph->cuda_real_edge, scan_edge, (cnvtxs+1)* sizeof(int), hipMemcpyHostToDevice);
  
  hipMalloc((void**)&cgraph->cuda_vwgt, cnvtxs*sizeof(int));  
  hipMalloc((void**)&graph->cuda_js, cnvtxs*sizeof(int));

  hipDeviceSynchronize();
  gettimeofday(&begin_initcudajs,NULL);
  initcudajs<<<cnvtxs/32+1,32>>>(graph->cuda_js,graph->cuda_cnvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_initcudajs,NULL);
  sinitcudajs += (end_initcudajs.tv_sec - begin_initcudajs.tv_sec) * 1000 + (end_initcudajs.tv_usec - begin_initcudajs.tv_usec) / 1000.0;
  
  hipDeviceSynchronize();
  gettimeofday(&begin_find_cnvtxsedge_original,NULL);
  find_cnvtxsedge_original<<<nvtxs,32>>>(graph->cuda_real_edge,graph->cuda_real_nvtxs,graph->cuda_xadj,graph->cuda_match,\
    graph->cuda_adjncy,graph->cuda_scan_nedges_original,graph->cuda_cmap,cgraph->cuda_vwgt,graph->cuda_vwgt,graph->cuda_js,\
    graph->cuda_scan_cnedges_original,graph->cuda_scan_adjwgt_original,graph->cuda_adjwgt);
  hipDeviceSynchronize();
  gettimeofday(&end_find_cnvtxsedge_original,NULL);
  sfind_cnvtxsedge_original += (end_find_cnvtxsedge_original.tv_sec - begin_find_cnvtxsedge_original.tv_sec) * 1000 + (end_find_cnvtxsedge_original.tv_usec - begin_find_cnvtxsedge_original.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_bb_segsort,NULL);
  bb_segsort(graph->cuda_scan_cnedges_original, graph->cuda_scan_adjwgt_original, graph->nedges, graph->cuda_real_edge, cnvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_bb_segsort,NULL);
  sbb_segsort += (end_bb_segsort.tv_sec - begin_bb_segsort.tv_sec) * 1000 + (end_bb_segsort.tv_usec - begin_bb_segsort.tv_usec) / 1000.0;

  int *temp_scan;
  hipMalloc((void**)&temp_scan, nedges*sizeof(int));

  hipDeviceSynchronize();
  gettimeofday(&begin_Sort_cnedges_part1,NULL);
  Sort_cnedges2_part1<<<cnvtxs,32>>>(graph->cuda_scan_cnedges_original,graph->cuda_real_edge,graph->cuda_real_nvtxs,\
    graph->cuda_cmap,temp_scan);
  hipDeviceSynchronize();
  gettimeofday(&end_Sort_cnedges_part1,NULL);
  sSort_cnedges_part1 += (end_Sort_cnedges_part1.tv_sec - begin_Sort_cnedges_part1.tv_sec) * 1000 + (end_Sort_cnedges_part1.tv_usec - begin_Sort_cnedges_part1.tv_usec) / 1000.0;
  
  hipDeviceSynchronize();
  gettimeofday(&begin_inclusive_scan,NULL);
  thrust::device_ptr<int> ccscan = thrust::device_pointer_cast<int>(temp_scan);
  thrust::inclusive_scan(ccscan,ccscan+nedges,ccscan);
  hipDeviceSynchronize();
  gettimeofday(&end_inclusive_scan,NULL);
  sinclusive_scan += (end_inclusive_scan.tv_sec - begin_inclusive_scan.tv_sec) * 1000 + (end_inclusive_scan.tv_usec - begin_inclusive_scan.tv_usec) / 1000.0;

  hipMalloc((void**)&cgraph->cuda_xadj, (cnvtxs+1)*sizeof(int));

  hipDeviceSynchronize();
  gettimeofday(&begin_Sort_cnedges_part2,NULL);
  Sort_cnedges2_part2<<<cnvtxs/32+1,32>>>(graph->cuda_real_edge,graph->cuda_real_nvtxs,\
    graph->cuda_cmap,temp_scan,cgraph->cuda_xadj,graph->cuda_cnvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_Sort_cnedges_part2,NULL);
  sSort_cnedges_part2 += (end_Sort_cnedges_part2.tv_sec - begin_Sort_cnedges_part2.tv_sec) * 1000 + (end_Sort_cnedges_part2.tv_usec - begin_Sort_cnedges_part2.tv_usec) / 1000.0;
  
  int *cxadj=(int *)malloc(sizeof(int)*(cnvtxs+1));
  hipMemcpy(cxadj, cgraph->cuda_xadj, (cnvtxs+1)* sizeof(int), hipMemcpyDeviceToHost); 

  cgraph->nvtxs=cnvtxs;
  cgraph->nedges=cxadj[cnvtxs];

  hipMalloc((void**)&cgraph->cuda_adjncy,   cgraph->nedges*sizeof(int));
  hipMalloc((void**)&cgraph->cuda_adjwgt,   cgraph->nedges*sizeof(int));

  int *ccc;
  hipMalloc((void**)&ccc,sizeof(int));
  hipMemcpy(ccc, &cgraph->nedges, sizeof(int), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  gettimeofday(&begin_Sort_cnedges_part2_5,NULL);
  Sort_cnedges2_part2_5<<<(cgraph->nedges/32)+1,32>>>(cgraph->cuda_adjwgt,cgraph->cuda_adjncy,ccc);
  hipDeviceSynchronize();
  gettimeofday(&end_Sort_cnedges_part2_5,NULL);
  sSort_cnedges_part2_5 += (end_Sort_cnedges_part2_5.tv_sec - begin_Sort_cnedges_part2_5.tv_sec) * 1000 + (end_Sort_cnedges_part2_5.tv_usec - begin_Sort_cnedges_part2_5.tv_usec) / 1000.0;

  //累加 权重 点3：5+1=6
  hipDeviceSynchronize();
  gettimeofday(&begin_Sort_cnedges_part3,NULL);
  Sort_cnedges2_part3<<<cnvtxs,32>>>(graph->cuda_scan_cnedges_original,graph->cuda_real_edge,graph->cuda_real_nvtxs,\
    graph->cuda_cmap,graph->cuda_scan_adjwgt_original,temp_scan,\
    cgraph->cuda_adjncy,cgraph->cuda_adjwgt);
  hipDeviceSynchronize();
  gettimeofday(&end_Sort_cnedges_part3,NULL);
  sSort_cnedges_part3 += (end_Sort_cnedges_part3.tv_sec - begin_Sort_cnedges_part3.tv_sec) * 1000 + (end_Sort_cnedges_part3.tv_usec - begin_Sort_cnedges_part3.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  hipFree(ccc);
  hipFree(temp_scan);

  cgraph->tvwgt[0]=graph->tvwgt[0];   

  if(level!=0){
    cuMetis_free_coarsen(graph);
  }

}



/*CUDA-init match array*/
__global__ void initcuda_match(int *cuda_match,int *a)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
  cuda_match[ii]=-1;
  }
}



/*CUDA-hem matching*/
__global__ void cuda_hem(int *cuda_nvtxs, int *match, int *xadj, int *vwgt,\
int *adjwgt, int *adjncy, int *maxvwgt)
{
  int pi;
  int ii;
  int i,j,k,maxidx,maxwgt;
  ii=blockIdx.x*blockDim.x+threadIdx.x;
  int b_start,b_end;
  int tt=1024;
  int nvtxs=cuda_nvtxs[0];

  if(nvtxs%tt==0){
    b_start=ii*(nvtxs/tt);
    b_end=(ii+1)*(nvtxs/tt);
  }
  else{
    int b=nvtxs/tt;
    int a=b+1;
    int x=nvtxs-b*tt;

    if(ii<x){
      b_start=ii*a;
      b_end=(ii+1)*a;
    }
    else{
      b_start=x*a+(ii-x)*b;
      b_end=x*a+(ii+1-x)*b;
    }
  }

  for(pi=b_start;pi<b_end;pi++){
    i=pi;

    if(match[i]==-1){  
      maxidx=i;                                                                               
      maxwgt=-1;       

      for(j=xadj[i];j<xadj[i+1];j++){
        k=adjncy[j];

        if(match[k]==-1&&maxwgt<adjwgt[j]&&vwgt[i]+vwgt[k]<=maxvwgt[0]){
          maxidx=k;
          maxwgt=adjwgt[j];
        }  
        if(maxidx==i&&3*vwgt[i]<maxvwgt[0]){ 
          maxidx = -1;
        }
      }
      if(maxidx!=-1){    
        match[i] = maxidx;  
        atomicExch(&match[maxidx],i);                                 
      }
    }
  }
}


//edge
__global__ void cuda_shem1(int *nvtxs, int *xadj, int *adjwgt, int *match, int *adjncy, int *vwgt, int *maxvwgt)
{
  int ii = blockIdx.x;
  int j, k, maxidx, maxwgt;

  // do
  // {
  //    __threadfence_block();
  //    __threadfence();
  // }while()
  if(threadIdx.x == 0)
  {
    maxidx = ii;                                                                               
    maxwgt = -1; 

    int cnt = xadj[ii + 1] - xadj[ii];

    do
    {
      if(cnt == 0) break;
      // printf("maxvwgt=%d\n",maxvwgt[0]);
      for(j = xadj[ii];j < xadj[ii + 1];++j)
      {
        k = adjncy[j];
        if (match[k] < 0 && maxwgt < adjwgt[j] && vwgt[ii]+vwgt[k] <= maxvwgt[0]) 
        {
          maxidx = k;
          maxwgt = adjwgt[j];
        }
        if (maxidx == ii && 3 * vwgt[ii] < maxvwgt[0]) 
        { 
          maxidx = -1;
        }
      }
      /*if (maxidx != -1) 
      {
        if(match[ii] == -1) match[ii] = maxidx;

        __syncthreads();

        if(match[maxidx] == ii) break;
        else cnt--;
      }
      else break;*/
      // printf("ii=%d maxidx=%d\n",ii,maxidx);
      if(maxidx != -1) 
      {
        __threadfence();
        // if(atomicCAS(&match[ii],-1,maxidx))
        if(match[ii] == -1)
        {
          // atomicExch(&match[ii],maxidx);
          __threadfence();
          // if(atomicCAS(&match[maxidx],-1,ii)) 
          if(match[maxidx] == -1)
          {
            match[ii] = maxidx;
            match[maxidx] = ii;
            // atomicExch(&match[ii],maxidx);
            // atomicExch(&match[maxidx],ii);
            // printf("--ii=%d match[%d]=%d\n",ii,ii,match[ii]);
            break;
          }
          else if(match[maxidx] != ii) 
          {
            // atomicExch(&match[ii],-1); 
            cnt--;
          }
          else break;
        } 
        else break;
      }
      else break;
    }while(match[ii] == -1);

    // printf("match[%d]=%d\n",ii,match[ii]);
  }
}



__global__ void cuda_shem2(int *nvtxs, int *xadj, int *adjwgt, int *match, int *adjncy, int *vwgt, int *maxvwgt)
{
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  int j, k, maxidx, maxwgt;

  if(ii < nvtxs[0])
  {
    if(3 * vwgt[ii] < maxvwgt[0])
    {
      for(j = xadj[ii + 1] - 1;j >= xadj[ii];--j)
      {
        k = adjncy[j];

        if(match[ii] != -1) break;
        else if(match[k] == ii)
        {
          match[ii] = k;
          break;
        }
        else if(match[k] != -1)
          continue;
        else if(vwgt[ii]+vwgt[k] > maxvwgt[0])
          continue;
        else
        {
          match[ii] = k;
          match[k] = ii;
          break;
        }
      }
    }
  }
}



/*CUDA-set conflict array*/
__global__ void cuda_cleanv(int *match, int *s, int *a)
{
  int pi,u;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<a[0]){
    s[pi]=1;

    if(match[pi]!=-1){
      u=match[pi];

      if(match[u]!=pi){
        s[pi]=0;
      }
    }
  } 
} 



/*CUDA-find cgraph vertex part1-remark the match array by s*/
__global__ void findc1(int *cuda_match, int *cuda_cmap, int *cuda_nvtxs, int *s)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<cuda_nvtxs[0]){
    if(s[pi]==0||cuda_match[pi] == -1)
    cuda_match[pi]=pi;
  }
}


/*CUDA-find cgraph vertex part2-make sure the pair small label vertex*/
__global__ void findc2(int *cuda_match, int *cuda_cmap, int *cuda_nvtxs, int *s)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<cuda_nvtxs[0]){
    if(pi<=cuda_match[pi]){
      cuda_cmap[pi]=1;
    }
    else{
      cuda_cmap[pi]=0;
    }
  }
}


/*CUDA-find cgraph vertex part2.5-init array*/
__global__ void findc2_5(int *cuda_temp, int *cuda_cmap, int *nvtxs)
{
  int i=nvtxs[0]-1;
  cuda_temp[0]=cuda_cmap[i];
}



/*CUDA-find cgraph vertex part3-array reduce 1*/
__global__ void findc3(int *cuda_match, int *cuda_cmap, int *cuda_nvtxs, int *s)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;
  if(pi<cuda_nvtxs[0]){
    cuda_cmap[pi]--;
  }
}


/*CUDA-find cgraph vertex part4-make sure vertex pair real rdge*/
__global__ void findc4(int *cuda_match, int *real, int *cmap, int *cnedges, int *xadj, int *nvtxs)
{
  int pi;
  int u;

  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<nvtxs[0]){
    if(pi>cuda_match[pi]){
      cmap[pi]=cmap[cuda_match[pi]];
    }
    else{
      u=cuda_match[pi];
      real[cmap[pi]]=pi;

      if(u!=pi){
        cnedges[cmap[pi]]=(xadj[pi+1]-xadj[pi])+(xadj[u+1]-xadj[u]);
      }
      else{
        cnedges[cmap[pi]]=(xadj[pi+1]-xadj[pi]);
      }
    }
  }
}



/*Get gpu graph matching params by hem*/
int cuMetis_gpu_match(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph, int level)
{
  hipDeviceSynchronize();
  gettimeofday(&begin_part_match,NULL);

  int nvtxs  = graph->nvtxs;
  int nedges = graph->nedges;

  hipDeviceSynchronize();
  gettimeofday(&begin_initcuda_match,NULL);
  initcuda_match<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_nvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_initcuda_match,NULL);
  sinitcuda_match += (end_initcuda_match.tv_sec - begin_initcuda_match.tv_sec) * 1000 + (end_initcuda_match.tv_usec - begin_initcuda_match.tv_usec) / 1000.0;

  hipMemcpy(  graph->cuda_maxvwgt, cuMetis_admin->maxvwgt, sizeof(int), hipMemcpyHostToDevice); 

  hipDeviceSynchronize();
  gettimeofday(&begin_cuda_match,NULL);
  // cuda_hem<<<1024,1>>>(graph->cuda_nvtxs,graph->cuda_match,graph->cuda_xadj,\
  //   graph-> cuda_vwgt,graph->cuda_adjwgt,graph->cuda_adjncy,graph->cuda_maxvwgt);
  // cuda_shem1<<<nvtxs,1>>>(graph->cuda_nvtxs,graph->cuda_xadj,graph->cuda_adjwgt,\
  //   graph->cuda_match,graph->cuda_adjncy,graph-> cuda_vwgt,graph->cuda_maxvwgt);
  
  bb_segsort(graph->cuda_adjwgt,graph->cuda_adjncy,nedges,graph->cuda_xadj,nvtxs);

  cuda_shem2<<<nvtxs/32+1,32>>>(graph->cuda_nvtxs,graph->cuda_xadj,graph->cuda_adjwgt,\
    graph->cuda_match,graph->cuda_adjncy,graph-> cuda_vwgt,graph->cuda_maxvwgt);

  hipDeviceSynchronize();
  gettimeofday(&end_cuda_match,NULL);
  scuda_match += (end_cuda_match.tv_sec - begin_cuda_match.tv_sec) * 1000 + (end_cuda_match.tv_usec - begin_cuda_match.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_cuda_cleanv,NULL);
  cuda_cleanv<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_s,graph->cuda_nvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_cuda_cleanv,NULL);
  scuda_cleanv += (end_cuda_cleanv.tv_sec - begin_cuda_cleanv.tv_sec) * 1000 + (end_cuda_cleanv.tv_usec - begin_cuda_cleanv.tv_usec) / 1000.0;

  int cnvtxs=0;

  hipDeviceSynchronize();
  gettimeofday(&begin_findc1,NULL);
  findc1<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_cmap,graph->cuda_nvtxs,graph->cuda_s);
  hipDeviceSynchronize();
  gettimeofday(&end_findc1,NULL);
  sfindc1 += (end_findc1.tv_sec - begin_findc1.tv_sec) * 1000 + (end_findc1.tv_usec - begin_findc1.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_findc2,NULL);
  findc2<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_cmap,graph->cuda_nvtxs,graph->cuda_s);
  hipDeviceSynchronize();
  gettimeofday(&end_findc2,NULL);
  sfindc2 += (end_findc2.tv_sec - begin_findc2.tv_sec) * 1000 + (end_findc2.tv_usec - begin_findc2.tv_usec) / 1000.0;

  thrust::device_ptr<int> cscan = thrust::device_pointer_cast<int>(graph->cuda_cmap);
  hipDeviceSynchronize();
  gettimeofday(&begin_inclusive_scan2,NULL);
  thrust::inclusive_scan(cscan,cscan+nvtxs,cscan);
  hipDeviceSynchronize();
  gettimeofday(&end_inclusive_scan2,NULL);
  sinclusive_scan2 += (end_inclusive_scan2.tv_sec - begin_inclusive_scan2.tv_sec) * 1000 + (end_inclusive_scan2.tv_usec - begin_inclusive_scan2.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_findc2_5,NULL);
  findc2_5<<<1,1>>>(graph->cuda_cnvtxs,graph->cuda_cmap,graph->cuda_nvtxs);
  hipDeviceSynchronize();
  gettimeofday(&end_findc2_5,NULL);
  sfindc2_5 += (end_findc2_5.tv_sec - begin_findc2_5.tv_sec) * 1000 + (end_findc2_5.tv_usec - begin_findc2_5.tv_usec) / 1000.0;
  
  hipMemcpy(  &cnvtxs,graph->cuda_cnvtxs,  sizeof(int), hipMemcpyDeviceToHost);
  
  int *cpu_scan_edge=(int *)malloc(sizeof(int)*(cnvtxs+1));
  hipMalloc((void**)&graph->cuda_real_nvtxs,  cnvtxs*sizeof(int));
  hipMalloc((void**)&graph->cuda_real_edge,  (cnvtxs+1)*sizeof(int));

  hipDeviceSynchronize();
  gettimeofday(&begin_findc3,NULL);
  findc3<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_cmap,graph->cuda_nvtxs,graph->cuda_s);
  hipDeviceSynchronize();
  gettimeofday(&end_findc3,NULL);
  sfindc3 += (end_findc3.tv_sec - begin_findc3.tv_sec) * 1000 + (end_findc3.tv_usec - begin_findc3.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_findc4,NULL);
  findc4<<<nvtxs/32+1,32>>>(graph->cuda_match,graph->cuda_real_nvtxs,graph->cuda_cmap,graph->cuda_real_edge,graph->cuda_xadj,graph->cuda_nvtxs);
  hipDeviceSynchronize();//预估粗点临界边
  gettimeofday(&end_findc4,NULL);
  sfindc4 += (end_findc4.tv_sec - begin_findc4.tv_sec) * 1000 + (end_findc4.tv_usec - begin_findc4.tv_usec) / 1000.0;
  hipMemcpy(  cpu_scan_edge, graph->cuda_real_edge,  (cnvtxs+1)*sizeof(int), hipMemcpyDeviceToHost);
  
  hipDeviceSynchronize();
  gettimeofday(&end_part_match,NULL);
  part_match += (end_part_match.tv_sec - begin_part_match.tv_sec) * 1000 + (end_part_match.tv_usec - begin_part_match.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_part_contract,NULL);

  cuMetis_gpu_create_cgraph(cuMetis_admin, graph, cnvtxs, level,cpu_scan_edge);  
  
  hipDeviceSynchronize();
  gettimeofday(&end_part_contract,NULL);
  part_contract += (end_part_contract.tv_sec - begin_part_contract.tv_sec) * 1000 + (end_part_contract.tv_usec - begin_part_contract.tv_usec) / 1000.0;
  
  return cnvtxs;

}


void cuMetis_memcpy_coarsentoinit(cuMetis_graph_t *graph)
{
  int nvtxs=graph->nvtxs;
  int nedges=graph->nedges;
  graph->vwgt=(int *)malloc(sizeof(int)*nvtxs); 
  graph->adjncy=(int *)malloc(sizeof(int)*nedges);
  graph->adjwgt=(int *)malloc(sizeof(int)*nedges);
  hipMemcpy(  graph->xadj, graph->cuda_xadj , (nvtxs+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(  graph->vwgt, graph->cuda_vwgt , (nvtxs)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(  graph->adjncy, graph->cuda_adjncy , (nedges)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(  graph->adjwgt, graph->cuda_adjwgt , (nedges)*sizeof(int), hipMemcpyDeviceToHost);
}


/*Gpu multilevel coarsen*/
cuMetis_graph_t *cuMetis_coarsen(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int level=0;

  cuMetis_admin->maxvwgt[0]=1.5*graph->tvwgt[0]/cuMetis_admin->Coarsen_threshold; 
  
  do{  
    if(level!=0){
      cuMetis_malloc_coarseninfo(cuMetis_admin,graph);
    }
    hipDeviceSynchronize();
    gettimeofday(&begin_sCoarsen,NULL);
    cuMetis_gpu_match(cuMetis_admin,graph,level);   
    gettimeofday(&end_sCoarsen,NULL);
    sCoarsen += (end_sCoarsen.tv_sec - begin_sCoarsen.tv_sec) * 1000 + (end_sCoarsen.tv_usec - begin_sCoarsen.tv_usec) / 1000.0;

    graph = graph->coarser;   
    level++;       

  }while(graph->nvtxs>cuMetis_admin->Coarsen_threshold&&\
      graph->nvtxs<0.75*graph->finer->nvtxs&&\
      graph->nedges>graph->nvtxs/2); 

  cuMetis_memcpy_coarsentoinit(graph);  

  return graph;
}


/*Cpu multilevel coarsen*/
cuMetis_graph_t *cuMetis_cpu_coarsen(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int level=1;

  cuMetis_admin->maxvwgt[0] = 1.5*graph->tvwgt[0]/cuMetis_admin->Coarsen_threshold;
 
  do{
    if(graph->cmap==NULL){
      graph->cmap=(int*)malloc(sizeof(int)*(graph->nvtxs));
    }

    cuMetis_cpu_match (cuMetis_admin,graph,level);

    graph = graph->coarser;
    level++;

  }while(graph->nvtxs > cuMetis_admin->Coarsen_threshold && 
      graph->nvtxs < 0.75*graph->finer->nvtxs && 
      graph->nedges > graph->nvtxs/2);

  return graph;
}



/*Malloc cpu 2way-refine params*/
void cuMetis_allocate_cpu_2waymem(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int nvtxs;
  nvtxs = graph->nvtxs;

  graph->pwgts=(int*)malloc(2*sizeof(int));
  graph->where=(int*)malloc(nvtxs*sizeof(int));
  graph->bndptr=(int*)malloc(nvtxs*sizeof(int));
  graph->bndlist=(int*)malloc(nvtxs*sizeof(int));
  graph->id=(int*)malloc(nvtxs*sizeof(int));
  graph->ed=(int*)malloc(nvtxs*sizeof(int));
}



/*Compute cpu 2way-refine params*/
void cuMetis_compute_cpu_2wayparam(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int i,j,nvtxs,nbnd,mincut,istart,iend,tid,ted,me;
  int *xadj,*vwgt,*adjncy,*adjwgt,*pwgts;
  int *where,*bndptr,*bndlist,*id,*ed;

  nvtxs= graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  where=graph->where;
  id=graph->id;
  ed=graph->ed;

  pwgts=cuMetis_int_set_value(2,0,graph->pwgts);
  bndptr=cuMetis_int_set_value(nvtxs,-1,graph->bndptr);

  bndlist=graph->bndlist;

  for(i=0;i<nvtxs;i++){
    pwgts[where[i]] += vwgt[i];
  }

  for(nbnd=0,mincut=0,i=0;i<nvtxs;i++){
    istart=xadj[i];
    iend=xadj[i+1];
    me=where[i];
    tid=ted=0;

    for(j=istart;j<iend;j++){
      if(me==where[adjncy[j]]){
        tid+=adjwgt[j];
      }
      else{
        ted+=adjwgt[j];
      }
    }

    id[i]=tid;
    ed[i]=ted;

    if(ted>0||istart==iend){
      cuMetis_listinsert(nbnd,bndlist,bndptr,i);
      mincut+=ted;
    }
  }

  graph->mincut=mincut/2;
  graph->nbnd=nbnd; 

}




/*Compute cpu imbalance params*/
float cuMetis_compute_cpu_imbal(cuMetis_graph_t *graph, int nparts, \
float *part_balance, float *ubvec)
{
  int j,*pwgts;
  float max,cur;
  pwgts=graph->pwgts;
  max=-1.0;

  for(j=0;j<nparts;j++){
    cur=pwgts[j]*part_balance[j]-ubvec[0];

    if(cur>max){
      max=cur;
    }
  }

  return max;
}




/*Init queue */
 void cuMetis_queue_init(cuMetis_queue_t *queue, size_t maxnodes)
{
  int i;
  queue->nnodes=0;
  queue->maxnodes=maxnodes;
  queue->heap=(cuMetis_rkv_t*)malloc(sizeof(cuMetis_rkv_t)*maxnodes);
  queue->locator=(ssize_t*)malloc(sizeof(ssize_t)*maxnodes);

  for(i=0;i<maxnodes;i++){
    queue->locator[i]=-1;
  }

}



/*Create queue*/
cuMetis_queue_t *cuMetis_queue_create(size_t maxnodes)
{
  cuMetis_queue_t *queue; 
  queue = (cuMetis_queue_t *)malloc(sizeof(cuMetis_queue_t));

  cuMetis_queue_init(queue, maxnodes);

  return queue;
}



/*Insert node to queue*/
int cuMetis_queue_insert(cuMetis_queue_t *queue, int node, int key)
{
  ssize_t i,j;
  ssize_t *locator=queue->locator;
  cuMetis_rkv_t *heap=queue->heap;
  i = queue->nnodes++;

  while(i>0){
    j=(i-1)>>1;

    if(M_GT_N(key,heap[j].key)){
      heap[i]=heap[j];
      locator[heap[i].val]=i;
      i=j;
    }
    else
      break;
  }

  heap[i].key=key;
  heap[i].val=node;
  locator[node]=i;

  return 0;

}



/*Get top of queue*/
int cuMetis_queue_top(cuMetis_queue_t *queue)
{
  ssize_t i, j;
  ssize_t *locator;
  cuMetis_rkv_t *heap;

  int vtx, node;
  float key;

  if (queue->nnodes==0){
    return -1;
  }

  queue->nnodes--;
  heap=queue->heap;
  locator=queue->locator;
  vtx=heap[0].val;
  locator[vtx]=-1;

  if ((i=queue->nnodes)>0){
    key=heap[i].key;
    node=heap[i].val;
    i=0;

    while((j=2*i+1)<queue->nnodes){
      if(M_GT_N(heap[j].key,key)){
        if(j+1 < queue->nnodes&&M_GT_N(heap[j+1].key,heap[j].key)){
          j=j+1;
        }

        heap[i]=heap[j];
        locator[heap[i].val]=i;
        i=j;
      }
      else if(j+1<queue->nnodes&&M_GT_N(heap[j+1].key,key)){
        j=j+1;
        heap[i]=heap[j];
        locator[heap[i].val]=i;
        i=j;
      }
      else
        break;
    }

    heap[i].key=key;
    heap[i].val=node;
    locator[node]=i;

  }

  return vtx;

}


/*Delete node of queue*/
int cuMetis_queue_delete(cuMetis_queue_t *queue, int node)
{
  ssize_t i, j, nnodes;
  float newkey, oldkey;
  ssize_t *locator=queue->locator;

  cuMetis_rkv_t *heap=queue->heap;

  i=locator[node];
  locator[node]=-1;

  if(--queue->nnodes>0&&heap[queue->nnodes].val!=node) {
    node=heap[queue->nnodes].val;
    newkey=heap[queue->nnodes].key;
    oldkey=heap[i].key;

    if(M_GT_N(newkey,oldkey)){ 
      while(i>0){
        j=(i-1)>>1;

        if(M_GT_N(newkey,heap[j].key)){
          heap[i]=heap[j];
          locator[heap[i].val]=i;
          i=j;
        }
        else
          break;
      }
    }
    else{ 
      nnodes=queue->nnodes;

      while((j=(i<<1)+1)<nnodes){
        if(M_GT_N(heap[j].key,newkey)){
          if(j+1<nnodes&&M_GT_N(heap[j+1].key,heap[j].key)){
            j++;
          }

          heap[i]=heap[j];
          locator[heap[i].val]=i;
          i=j;
        }
        else if(j+1<nnodes&&M_GT_N(heap[j+1].key,newkey)){
          j++;
          heap[i]=heap[j];
          locator[heap[i].val]=i;
          i=j;
        }
        else
          break;
      }
    }

    heap[i].key=newkey;
    heap[i].val=node;
    locator[node]=i;

  }

  return 0;
}



/*Update queue node key*/
void cuMetis_queue_update(cuMetis_queue_t *queue, int node, int newkey)
{
  ssize_t i, j, nnodes;
  float oldkey;
  ssize_t *locator=queue->locator;

  cuMetis_rkv_t *heap=queue->heap;
  oldkey=heap[locator[node]].key;
  i=locator[node];

  if(M_GT_N(newkey,oldkey)){ 
    while(i>0){
      j=(i-1)>>1;

      if(M_GT_N(newkey,heap[j].key)){
        heap[i]=heap[j];
        locator[heap[i].val]=i;
        i=j;
      }
      else
        break;
    }
  }
  else{ 
    nnodes = queue->nnodes;

    while((j=(i<<1)+1)<nnodes){
      if(M_GT_N(heap[j].key,newkey)){
        if(j+1<nnodes&&M_GT_N(heap[j+1].key,heap[j].key)){
          j++;
        }

        heap[i]=heap[j];
        locator[heap[i].val]=i;
        i=j;
      }
      else if(j+1<nnodes&&M_GT_N(heap[j+1].key,newkey)){
        j++;
        heap[i]=heap[j];
        locator[heap[i].val]=i;
        i=j;
      }
      else
        break;
    }
  }

  heap[i].key=newkey;
  heap[i].val=node;
  locator[node]=i;
  return;

}



/*Free queue*/
void cuMetis_queue_free(cuMetis_queue_t *queue)
{
  if(queue == NULL) return;

  free(queue->heap);
  free(queue->locator);

  queue->maxnodes = 0;

  free(queue);
}



/*Reset queue*/
void cuMetis_queue_reset(cuMetis_queue_t *queue)
{
  ssize_t i;
  ssize_t *locator=queue->locator;

  cuMetis_rkv_t *heap=queue->heap;

  for(i=queue->nnodes-1;i>=0;i--){
    locator[heap[i].val]=-1;
  }

  queue->nnodes=0;

}



/*Balance two partition by moving boundary vertex*/
void cuMetis_bndvertex_2way_bal(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph, float *ntpwgts)
{
  int i,ii,j,k,kwgt,nvtxs,nbnd,nswaps,from,to,temp;
  int *xadj,*vwgt,*adjncy,*adjwgt,*where,*id,*ed,*bndptr,*bndlist,*pwgts;
  int *moved,*perm;

  cuMetis_queue_t *queue;
  int higain,mincut,mindiff;
  int tpwgts[2];

  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  where=graph->where;
  id=graph->id;
  ed=graph->ed;
  pwgts=graph->pwgts;
  bndptr=graph->bndptr;
  bndlist=graph->bndlist;

  moved=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  perm=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);

  tpwgts[0]=graph->tvwgt[0]*ntpwgts[0];
  tpwgts[1]=graph->tvwgt[0]-tpwgts[0];
  mindiff=abs(tpwgts[0]-pwgts[0]);
  from=(pwgts[0]<tpwgts[0]?1:0);
  to=(from+1)%2;

  queue=cuMetis_queue_create(nvtxs);
  cuMetis_int_set_value(nvtxs,-1,moved);
  nbnd=graph->nbnd;
  cuMetis_int_randarrayofp(nbnd,perm,nbnd/5,1);

  for(ii=0;ii<nbnd;ii++){
    i=perm[ii];

    if(where[bndlist[i]]==from&&vwgt[bndlist[i]]<=mindiff){
      cuMetis_queue_insert(queue,bndlist[i],ed[bndlist[i]]-id[bndlist[i]]);
    }
  }

  mincut=graph->mincut;

  for(nswaps=0;nswaps<nvtxs;nswaps++) 
  {
    if((higain=cuMetis_queue_top(queue))==-1)
      break;
    if(pwgts[to]+vwgt[higain]>tpwgts[to])
      break;

    mincut-=(ed[higain]-id[higain]);
    cuMetis_add_sub(pwgts[to],pwgts[from],vwgt[higain]);

    where[higain]=to;
    moved[higain]=nswaps;
    cuMetis_swap(id[higain],ed[higain],temp);

    if(ed[higain]==0&&xadj[higain]<xadj[higain+1]){ 
      cuMetis_listdelete(nbnd,bndlist,bndptr,higain);
    }

    for(j=xadj[higain];j<xadj[higain+1];j++){
      k=adjncy[j];
      kwgt=(to==where[k]?adjwgt[j]:-adjwgt[j]);
      cuMetis_add_sub(id[k],ed[k],kwgt);

      if(bndptr[k]!=-1){ 
        if(ed[k]==0){ 
          cuMetis_listdelete(nbnd,bndlist,bndptr,k);

          if(moved[k]==-1&&where[k]==from&&vwgt[k]<=mindiff){ 
            cuMetis_queue_delete(queue,k);
          }
        }
        else{ 
          if(moved[k]==-1&&where[k]==from&&vwgt[k]<=mindiff){
            cuMetis_queue_update(queue,k,ed[k]-id[k]);
          }
        }
      }
      else{
        if(ed[k]>0){  
          cuMetis_listinsert(nbnd,bndlist,bndptr,k);

          if(moved[k]==-1&&where[k]==from&&vwgt[k]<=mindiff){ 
            cuMetis_queue_insert(queue,k,ed[k]-id[k]);
          }
        }
      }
    }
  }

  graph->mincut=mincut;
  graph->nbnd=nbnd;
  cuMetis_queue_free(queue);

}



/*Balance 2-way partition*/
void cuMetis_2way_bal(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph, float *ntpwgts)
{
  if(cuMetis_compute_cpu_imbal(graph,2,cuMetis_admin->part_balance,cuMetis_admin->ubfactors)<=0){ 
    return;
  }

  if(abs(ntpwgts[0]*graph->tvwgt[0]-graph->pwgts[0])<3*graph->tvwgt[0]/graph->nvtxs){
    return;
  }

  cuMetis_bndvertex_2way_bal(cuMetis_admin,graph,ntpwgts);
}



/*Cpu graph refine two partitions*/
void cuMetis_cpu_2way_refine(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph, float *ntpwgts, int iteration_num)
{
  int i,ii,j,k,kwgt,nvtxs,nbnd,nswaps,from,to,pass,limit,temp;
  int *xadj,*vwgt,*adjncy,*adjwgt,*where,*id,*ed,*bndptr,*bndlist,*pwgts;
  int *moved,*swaps,*perm;

  cuMetis_queue_t *queues[2];
  int higain,mincut, mindiff,origdiff,initcut,newcut,mincutorder,avgvwgt;
  int tpwgts[2];

  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  where=graph->where;
  id=graph->id;
  ed=graph->ed;
  pwgts=graph->pwgts;
  bndptr=graph->bndptr;
  bndlist=graph->bndlist;

  moved=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  swaps=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  perm=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);

  tpwgts[0]=graph->tvwgt[0]*ntpwgts[0];
  tpwgts[1]=graph->tvwgt[0]-tpwgts[0];

  limit=cuMetis_min(cuMetis_max(0.01*nvtxs,15),100);
  avgvwgt=cuMetis_min((pwgts[0]+pwgts[1])/20,2*(pwgts[0]+pwgts[1])/nvtxs);

  queues[0]=cuMetis_queue_create(nvtxs);
  queues[1]=cuMetis_queue_create(nvtxs);

  origdiff=abs(tpwgts[0]-pwgts[0]);
  cuMetis_int_set_value(nvtxs,-1,moved);

  for(pass=0;pass<iteration_num;pass++){ 
    cuMetis_queue_reset(queues[0]);
    cuMetis_queue_reset(queues[1]);

    mincutorder=-1;
    newcut=mincut=initcut=graph->mincut;
    mindiff=abs(tpwgts[0]-pwgts[0]);
    nbnd=graph->nbnd;
    cuMetis_int_randarrayofp(nbnd,perm,nbnd,1); 

    for(ii=0;ii<nbnd;ii++){
      i=perm[ii];
      cuMetis_queue_insert(queues[where[bndlist[i]]],bndlist[i],ed[bndlist[i]]-id[bndlist[i]]);
    }       

    for(nswaps=0;nswaps<nvtxs;nswaps++){
      from=(tpwgts[0]-pwgts[0]<tpwgts[1]-pwgts[1]?0:1);
      to=(from+1)%2;

      if((higain=cuMetis_queue_top(queues[from]))==-1){
        break;
      }

      newcut-=(ed[higain]-id[higain]);
      cuMetis_add_sub(pwgts[to],pwgts[from],vwgt[higain]);

      if((newcut<mincut&&abs(tpwgts[0]-pwgts[0])<=origdiff+avgvwgt)|| 
          (newcut==mincut&&abs(tpwgts[0]-pwgts[0])<mindiff)){
        mincut=newcut;
        mindiff=abs(tpwgts[0]-pwgts[0]);
        mincutorder=nswaps;
      }
      else if(nswaps-mincutorder>limit){ 
        newcut+=(ed[higain]-id[higain]);
        cuMetis_add_sub(pwgts[from],pwgts[to],vwgt[higain]);
        break;
      }

      where[higain]=to;
      moved[higain]=nswaps;
      swaps[nswaps]=higain;

      cuMetis_swap(id[higain],ed[higain],temp);

      if(ed[higain]==0&&xadj[higain]<xadj[higain+1]){ 
        cuMetis_listdelete(nbnd,bndlist,bndptr,higain);
      }

      for(j=xadj[higain];j<xadj[higain+1];j++){
        k=adjncy[j];
        kwgt=(to==where[k]?adjwgt[j]:-adjwgt[j]);
        cuMetis_add_sub(id[k],ed[k],kwgt);

        if(bndptr[k]!=-1){ 
          if(ed[k]==0){ 
            cuMetis_listdelete(nbnd,bndlist,bndptr,k);
            
            if(moved[k]==-1){  
              cuMetis_queue_delete(queues[where[k]],k);
            }
          }
          else{ 
            if(moved[k]==-1){ 
              cuMetis_queue_update(queues[where[k]],k,ed[k]-id[k]);
            }
          }
        }
        else{
          if(ed[k]>0){  
            cuMetis_listinsert(nbnd,bndlist,bndptr,k);
            
            if(moved[k]==-1){ 
              cuMetis_queue_insert(queues[where[k]],k,ed[k]-id[k]);
            }
          }
        }
      }
    }

    for(i=0;i<nswaps;i++){
      moved[swaps[i]]=-1;  
    }

    for(nswaps--;nswaps>mincutorder;nswaps--){
      higain=swaps[nswaps];
      to=where[higain]=(where[higain]+1)%2;
      cuMetis_swap(id[higain],ed[higain],temp);

      if(ed[higain]==0&&bndptr[higain]!=-1&&xadj[higain]<xadj[higain+1]){
        cuMetis_listdelete(nbnd,bndlist,bndptr,higain);
      }
      else if(ed[higain]>0&&bndptr[higain]==-1){
        cuMetis_listinsert(nbnd,bndlist,bndptr,higain);
      }

      cuMetis_add_sub(pwgts[to],pwgts[(to+1)%2],vwgt[higain]);

      for(j=xadj[higain];j<xadj[higain+1];j++){
        k=adjncy[j];
        kwgt=(to==where[k]?adjwgt[j]:-adjwgt[j]);
        cuMetis_add_sub(id[k],ed[k],kwgt);

        if(bndptr[k]!=-1&&ed[k]==0){
          cuMetis_listdelete(nbnd,bndlist,bndptr,k);
        }
        if(bndptr[k]==-1&&ed[k]>0){
          cuMetis_listinsert(nbnd,bndlist,bndptr,k);
        }
      }
    }

    graph->mincut=mincut;
    graph->nbnd=nbnd;

    // printf("pass=%d nvtxs=%d\n",pass,nvtxs);
    // printf("graph->mincut=%d\n\n",graph->mincut);

    if(mincutorder<=0||mincut==initcut){
      break;
    }

  }

  cuMetis_queue_free(queues[0]);
  cuMetis_queue_free(queues[1]);

}



/*Cpu growbisection algorithm*/
void cuMetis_cpu_growbisection(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, float *ntpwgts, int niparts)
{
  int i,j,k,nvtxs,dd,nleft,first,last,pwgts[2],oneminpwgt,onemaxpwgt, 
      bestcut=0,iter;

  int *xadj,*vwgt,*adjncy,*where;
  int *queue,*tra,*bestwhere;

  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;

  cuMetis_allocate_cpu_2waymem(cuMetis_admin,graph);

  where=graph->where;

  bestwhere=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  queue=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  tra=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);

  onemaxpwgt=cuMetis_admin->ubfactors[0]*graph->tvwgt[0]*ntpwgts[1];
  oneminpwgt=(1.0/cuMetis_admin->ubfactors[0])*graph->tvwgt[0]*ntpwgts[1]; 
  
  for (iter=0; iter<niparts; iter++){

    hipDeviceSynchronize();
    gettimeofday(&begin_part_bfs,NULL);
    
    cuMetis_int_set_value(nvtxs,1,where);
    cuMetis_int_set_value(nvtxs,0,tra);

    pwgts[1]=graph->tvwgt[0];
    pwgts[0]=0;
    queue[0]=cuMetis_int_randinrange(nvtxs);
    tra[queue[0]]=1;
    first=0; 
    last=1;
    nleft=nvtxs-1;
    dd=0;

    for(;;){
      if(first==last){ 
        if(nleft==0||dd){
          break;
        }

        k=cuMetis_int_randinrange(nleft);

        for(i=0;i<nvtxs;i++){
          if(tra[i]==0){
            if(k==0){
              break;
            }
            else{
              k--;
            }
          }
        }

        queue[0]=i;
        tra[i]=1;
        first=0; 
        last=1;
        nleft--;
      }

      i=queue[first++];

      if(pwgts[0]>0&&pwgts[1]-vwgt[i]<oneminpwgt){
        dd=1;
        continue;
      }

      where[i]=0;

      cuMetis_add_sub(pwgts[0],pwgts[1],vwgt[i]);

      if(pwgts[1]<=onemaxpwgt){
        break;
      }

      dd=0;

      for(j=xadj[i];j<xadj[i+1];j++){
        k=adjncy[j];

        if(tra[k]==0){
          queue[last++]=k;
          tra[k]=1;
          nleft--;
        }
      }
    }

    hipDeviceSynchronize();
    gettimeofday(&end_part_bfs,NULL);
    part_bfs += (end_part_bfs.tv_sec - begin_part_bfs.tv_sec) * 1000 + (end_part_bfs.tv_usec - begin_part_bfs.tv_usec) / 1000.0;

    cuMetis_compute_cpu_2wayparam(cuMetis_admin,graph);
    cuMetis_2way_bal(cuMetis_admin,graph,ntpwgts);

    hipDeviceSynchronize();
    gettimeofday(&begin_part_2refine,NULL);

    cuMetis_cpu_2way_refine(cuMetis_admin,graph,ntpwgts,cuMetis_admin->iteration_num);

    hipDeviceSynchronize();
    gettimeofday(&end_part_2refine,NULL);
    part_2refine += (end_part_2refine.tv_sec - begin_part_2refine.tv_sec) * 1000 + (end_part_2refine.tv_usec - begin_part_2refine.tv_usec) / 1000.0;
    
    if(iter==0||bestcut>graph->mincut){
      bestcut=graph->mincut;
      cuMetis_int_copy(nvtxs,where,bestwhere);
      
      if(bestcut==0){
        break;
      }
    }
  }

  graph->mincut=bestcut;
  cuMetis_int_copy(nvtxs,bestwhere,where);

}



/*Free graph params*/
void cuMetis_free_graph(cuMetis_graph_t **r_graph) 
{
  cuMetis_graph_t *graph;
  graph=*r_graph;

  free(graph->xadj);
  free(graph->vwgt);
  free(graph->adjncy);
  free(graph->adjwgt);
  free(graph->where);
  free(graph->pwgts);
  free(graph->id);
  free(graph->ed);
  free(graph->bndptr);
  free(graph->bndlist);
  free(graph->tvwgt);
  free(graph->tvwgt_reverse);
  free(graph->label);
  free(graph->cmap);
  free(graph);

  *r_graph = NULL;
}



/*Cpu graph 2-way projection*/
void cuMetis_2way_project(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int i,j,istart,iend,nvtxs,nbnd,me,tid,ted;
  int *xadj,*adjncy,*adjwgt;
  int *cmap,*where,*bndptr,*bndlist;
  int *cwhere,*cbndptr;
  int *id,*ed;

  cuMetis_graph_t *cgraph;
  cuMetis_allocate_cpu_2waymem(cuMetis_admin,graph);

  cgraph=graph->coarser;
  cwhere=cgraph->where;
  cbndptr=cgraph->bndptr;
  nvtxs=graph->nvtxs;
  cmap=graph->cmap;
  xadj=graph->xadj;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  where=graph->where;
  id=graph->id;
  ed=graph->ed;

  bndptr=cuMetis_int_set_value(nvtxs,-1,graph->bndptr);
  bndlist=graph->bndlist;

  for(i=0;i<nvtxs;i++){
    j=cmap[i];
    where[i]=cwhere[j];
    cmap[i]=cbndptr[j];
  }

  for(nbnd=0,i=0;i<nvtxs;i++){
    istart=xadj[i];
    iend=xadj[i+1];
    tid=ted=0;

    if(cmap[i]==-1){ 
      for(j=istart;j<iend;j++){
        tid+=adjwgt[j];
      }
    }
    else{ 
      me=where[i];

      for(j=istart;j<iend;j++){
        if(me==where[adjncy[j]]){
          tid += adjwgt[j];
        }
        else{
          ted+=adjwgt[j];
        }
      }
    }

    id[i]=tid;
    ed[i]=ted;

    if(ted>0||istart==iend){ 
      cuMetis_listinsert(nbnd,bndlist,bndptr,i);
    }

  }

  graph->mincut=cgraph->mincut;
  graph->nbnd=nbnd;

  cuMetis_int_copy(2,cgraph->pwgts,graph->pwgts);
  cuMetis_free_graph(&graph->coarser);
  graph->coarser=NULL;

}



/*Cpu refinement algorithm*/
void cuMetis_cpu_refinement(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *orggraph, cuMetis_graph_t *graph, float *tpwgts)
{
  cuMetis_compute_cpu_2wayparam(cuMetis_admin,graph);

  for(;;){
    cuMetis_2way_bal(cuMetis_admin,graph,tpwgts);

    hipDeviceSynchronize();
    gettimeofday(&begin_part_2refine,NULL);

    cuMetis_cpu_2way_refine(cuMetis_admin,graph,tpwgts,cuMetis_admin->iteration_num); 

    hipDeviceSynchronize();
    gettimeofday(&end_part_2refine,NULL);
    part_2refine += (end_part_2refine.tv_sec - begin_part_2refine.tv_sec) * 1000 + (end_part_2refine.tv_usec - begin_part_2refine.tv_usec) / 1000.0;
    
    if(graph==orggraph){
      break;
    }

    graph=graph->finer;

    hipDeviceSynchronize();
    gettimeofday(&begin_part_2map,NULL);

    cuMetis_2way_project(cuMetis_admin,graph);

    hipDeviceSynchronize();
    gettimeofday(&end_part_2map,NULL);
    part_2map += (end_part_2map.tv_sec - begin_part_2map.tv_sec) * 1000 + (end_part_2map.tv_usec - begin_part_2map.tv_usec) / 1000.0;
  }

}


/*Cpu multilevel bisection algorithm*/
int cuMetis_cpu_mlevelbisect(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, float *tpwgts)
{
  int niparts,bestobj=0,curobj=0,*bestwhere=NULL;
  cuMetis_graph_t *cgraph;

  cuMetis_compute_2way_balance(cuMetis_admin,graph,tpwgts);
  cgraph=cuMetis_cpu_coarsen(cuMetis_admin,graph);

  niparts=5;
  cuMetis_cpu_growbisection(cuMetis_admin,cgraph,tpwgts,niparts);

  cuMetis_cpu_refinement(cuMetis_admin,graph,cgraph,tpwgts);
 
  curobj=graph->mincut;
  bestobj=curobj;

  if(bestobj!=curobj){
    cuMetis_int_copy(graph->nvtxs,bestwhere,graph->where);
    cuMetis_compute_cpu_2wayparam(cuMetis_admin,graph);
  }

  return bestobj;
}



/*Set split graph params*/
cuMetis_graph_t *cuMetis_set_splitgraph(cuMetis_graph_t *graph, \
int snvtxs, int snedges)
{
  cuMetis_graph_t *sgraph;
  sgraph=cuMetis_create_cpu_graph();

  sgraph->nvtxs=snvtxs;
  sgraph->nedges=snedges;

  sgraph->xadj=(int*)malloc(sizeof(int)*(snvtxs+1));
  sgraph->vwgt=(int*)malloc(sizeof(int)*(snvtxs+1));
  sgraph->adjncy=(int*)malloc(sizeof(int)*(snedges));
  sgraph->adjwgt=(int*)malloc(sizeof(int)*(snedges));
  sgraph->label=(int*)malloc(sizeof(int)*(snvtxs));
  sgraph->tvwgt=(int*)malloc(sizeof(int));
  sgraph->tvwgt_reverse=(float*)malloc(sizeof(float));

  return sgraph;

}



/*Split graph to lgraph and rgraph*/
void cuMetis_splitgraph(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, cuMetis_graph_t **r_lgraph, cuMetis_graph_t **r_rgraph)
{
  int i,j,k,l,istart,iend,mypart,nvtxs,snvtxs[2],snedges[2];
  int *xadj,*vwgt,*adjncy,*adjwgt,*label,*where,*bndptr;
  int *sxadj[2],*svwgt[2],*sadjncy[2],*sadjwgt[2],*slabel[2];
  int *rename;
  int *temp_adjncy,*temp_adjwgt;

  cuMetis_graph_t *lgraph,*rgraph;

  nvtxs=graph->nvtxs;
  xadj=graph->xadj;
  vwgt=graph->vwgt;
  adjncy=graph->adjncy;
  adjwgt=graph->adjwgt;
  label=graph->label;
  where=graph->where;
  bndptr=graph->bndptr;

  rename=cuMetis_int_malloc_space(cuMetis_admin,nvtxs);
  snvtxs[0]=snvtxs[1]=snedges[0]=snedges[1]=0;

  for(i=0;i<nvtxs;i++){
    k=where[i];
    rename[i]=snvtxs[k]++;
    snedges[k]+=xadj[i+1]-xadj[i];
  }

  lgraph=cuMetis_set_splitgraph(graph,snvtxs[0],snedges[0]);
  sxadj[0]=lgraph->xadj;
  svwgt[0]=lgraph->vwgt;
  sadjncy[0]=lgraph->adjncy; 	
  sadjwgt[0]=lgraph->adjwgt; 
  slabel[0]=lgraph->label;

  rgraph=cuMetis_set_splitgraph(graph,snvtxs[1],snedges[1]);
  sxadj[1]=rgraph->xadj;
  svwgt[1]=rgraph->vwgt;
  sadjncy[1]=rgraph->adjncy; 	
  sadjwgt[1]=rgraph->adjwgt; 
  slabel[1]=rgraph->label;

  snvtxs[0]=snvtxs[1]=snedges[0]=snedges[1]=0;
  sxadj[0][0]=sxadj[1][0]=0;

  for(i=0;i<nvtxs;i++){
    mypart=where[i];
    istart=xadj[i];
    iend=xadj[i+1];

    if(bndptr[i]==-1){ 
      temp_adjncy=sadjncy[mypart]+snedges[mypart]-istart;
      temp_adjwgt=sadjwgt[mypart]+snedges[mypart]-istart;

      for(j=istart;j<iend;j++){
        temp_adjncy[j]=adjncy[j];
        temp_adjwgt[j]=adjwgt[j]; 
      }

      snedges[mypart]+=iend-istart;
    }
    else{
      temp_adjncy=sadjncy[mypart];
      temp_adjwgt=sadjwgt[mypart];
      l=snedges[mypart];

      for(j=istart;j<iend;j++){
        k=adjncy[j];
        
        if(where[k]==mypart){
          temp_adjncy[l]=k;
          temp_adjwgt[l++]=adjwgt[j]; 
        }
      }
      snedges[mypart]=l;
    }

    svwgt[mypart][snvtxs[mypart]]=vwgt[i];
    slabel[mypart][snvtxs[mypart]]=label[i];
    sxadj[mypart][++snvtxs[mypart]]=snedges[mypart];
  }

  for(mypart=0;mypart<2;mypart++){
    iend=sxadj[mypart][snvtxs[mypart]];
    temp_adjncy=sadjncy[mypart];

    for(i=0;i<iend;i++){ 
      temp_adjncy[i]=rename[temp_adjncy[i]];
    }
  }

  lgraph->nedges=snedges[0];
  rgraph->nedges=snedges[1];

  cuMetis_set_graph_tvwgt(lgraph);
  cuMetis_set_graph_tvwgt(rgraph);

  *r_lgraph=lgraph;
  *r_rgraph=rgraph;

}



/*Cpu Multilevel resursive bisection*/
int cuMetis_mlevel_rbbisection(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, int nparts, int *part, float *tpwgts, int fpart)
{
  int i,nvtxs,objval;
  int *label,*where;

  cuMetis_graph_t *lgraph,*rgraph;
  float wsum,*tpwgts2;

  if(graph->nvtxs==0){
    printf("****You are trying to partition too many parts!****\n");
    return 0;
  }

  nvtxs=graph->nvtxs;

  tpwgts2=cuMetis_float_malloc_space(cuMetis_admin);
  tpwgts2[0]=cuMetis_float_sum((nparts>>1),tpwgts);
  tpwgts2[1]=1.0-tpwgts2[0];

  objval=cuMetis_cpu_mlevelbisect(cuMetis_admin,graph,tpwgts2);
  
  label=graph->label;
  where=graph->where;

  for(i=0;i<nvtxs;i++){
    part[label[i]]=where[i]+fpart;
  }
  for(i=0;i<nvtxs;i++){
    part[label[i]]=where[i]+fpart;
  }

  if(nparts>2){ 
    hipDeviceSynchronize();
    gettimeofday(&begin_part_slipt,NULL);

    cuMetis_splitgraph(cuMetis_admin,graph,&lgraph,&rgraph);

    hipDeviceSynchronize();
    gettimeofday(&end_part_slipt,NULL);
    part_slipt += (end_part_slipt.tv_sec - begin_part_slipt.tv_sec) * 1000 + (end_part_slipt.tv_usec - begin_part_slipt.tv_usec) / 1000.0;
  }

  cuMetis_free_graph(&graph);

  wsum=cuMetis_float_sum((nparts>>1),tpwgts);
  
  cuMetis_tpwgts_rescale((nparts>>1),1.0/wsum,tpwgts);
  cuMetis_tpwgts_rescale(nparts-(nparts>>1),1.0/(1.0-wsum),tpwgts+(nparts>>1));
  
  if(nparts>3){
    objval+=cuMetis_mlevel_rbbisection(cuMetis_admin,lgraph,(nparts>>1),part,tpwgts,fpart);
    objval+=cuMetis_mlevel_rbbisection(cuMetis_admin,rgraph,nparts-(nparts>>1),part,tpwgts+(nparts>>1),fpart+(nparts>>1));
  }
  else if(nparts==3){
    cuMetis_free_graph(&lgraph);
    objval+=cuMetis_mlevel_rbbisection(cuMetis_admin,rgraph,nparts-(nparts>>1),part,tpwgts+(nparts>>1),fpart+(nparts>>1));
  }
  
  return objval;

}




/*Set kway balance params*/
void cuMetis_set_kway_bal(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph)
{
  int i,j;

  for(i=0;i<cuMetis_admin->nparts;i++){
    for(j=0;j<1;j++){
      cuMetis_admin->part_balance[i+j]=graph->tvwgt_reverse[j]/cuMetis_admin->tpwgts[i+j];
    }
  }
}



/*Cpu graph partition algorithm*/
int cuMetis_rbbisection(int *nvtxs, int *xadj, int *adjncy, int *vwgt,int *adjwgt, \
int *nparts, float *tpwgts, float *ubvec, int *objval, int *part)
{
  cuMetis_graph_t *graph;
  cuMetis_admin_t *cuMetis_admin;

  cuMetis_admin = cuMetis_set_graph_admin( *nparts, tpwgts, ubvec);

  graph = cuMetis_set_graph(cuMetis_admin, *nvtxs, xadj, adjncy, vwgt, adjwgt);

  cuMetis_allocatespace(cuMetis_admin, graph);           
  
  *objval = cuMetis_mlevel_rbbisection(cuMetis_admin, graph, *nparts, part, cuMetis_admin->tpwgts, 0);
  
  return 1;
 
}



/*CUDA-kway parjection*/
__global__ void projectback(int *where, int *cwhere, int *cmap, int *nvtxs)
{
  int pi;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<nvtxs[0]){
    where[pi]=cwhere[cmap[pi]];
  }
}


/*Kway parjection*/
void cuMetis_kway_project(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{       
  int nvtxs=graph->nvtxs;
  cuMetis_graph_t *cgraph; 

  cgraph=graph->coarser;
  
  projectback<<<nvtxs/32+1,32>>>(graph->cuda_where,cgraph->cuda_where,graph->cuda_cmap,graph->cuda_nvtxs);
}


/*Graph initial partition algorithm*/
void cuMetis_initialpartition(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph)
{
  int objval=0;
  int *bestwhere=NULL;
  float *ubvec=NULL;

  graph->where=(int *)malloc(sizeof(int)*graph->nvtxs);

  ubvec=(float*)malloc(sizeof(float));
  ubvec[0]=(float)pow(cuMetis_admin->ubfactors[0],1.0/log(cuMetis_admin->nparts));
  
  cuMetis_rbbisection(&graph->nvtxs,graph->xadj,graph->adjncy,graph->vwgt,graph->adjwgt, \
    &cuMetis_admin->nparts,cuMetis_admin->tpwgts,ubvec,&objval,graph->where);
  
  free(ubvec);
  free(bestwhere);
}



/*CUDA-init pwgts array*/
__global__ void initpwgts(int *cuda_pwgts, int *a)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    cuda_pwgts[ii]=0;
  }

}


/*CUDA-init pwgts array*/
__global__ void inittpwgts(float *tpwgts, float *temp, int *a)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    tpwgts[ii]=temp[0];
  }

}



/*Compute sum of pwgts*/
__global__ void Sumpwgts(int *cuda_pwgts, int *cuda_where, int *cuda_vwgt, int *nvtxs)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<nvtxs[0]){
    atomicAdd(&cuda_pwgts[cuda_where[ii]],cuda_vwgt[ii]);
  }
}


/*Malloc initial partition phase to refine phase params*/
void Mallocinit_refineinfo(cuMetis_admin_t *cuMetis_admin,\
cuMetis_graph_t *graph)
{
  int nvtxs=graph->nvtxs;
  int nparts=cuMetis_admin->nparts;

  hipMalloc((void**)&graph->cuda_where,nvtxs*sizeof(int));
  hipMemcpy(graph->cuda_where,graph->where,nvtxs*sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_bnd,nvtxs*sizeof(int));

  int num=0;

  hipMalloc((void**)&graph->cuda_bndnum,sizeof(int));
  hipMemcpy(graph->cuda_bndnum,&num,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_nparts,sizeof(int));
  hipMemcpy(graph->cuda_nparts,&nparts,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_pwgts,nparts*sizeof(int));

  initpwgts<<<nparts/32+1,32>>>(graph->cuda_pwgts,graph->cuda_nparts);

  hipMalloc((void**)&graph->cuda_nvtxs,sizeof(int));
  hipMemcpy(graph->cuda_nvtxs,&nvtxs,sizeof(int),hipMemcpyHostToDevice);

  Sumpwgts<<<nvtxs/32+1,32>>>(graph->cuda_pwgts,graph->cuda_where,graph->cuda_vwgt,graph->cuda_nvtxs);
  
  hipMalloc((void**)&graph->cuda_tvwgt,sizeof(int));
  hipMemcpy(graph->cuda_tvwgt,graph->tvwgt,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_tpwgts,nparts*sizeof(float));
  hipMalloc((void**)&graph->cuda_maxwgt,nparts*sizeof(int));
  hipMalloc((void**)&graph->cuda_minwgt,nparts*sizeof(int));

  float *temp;
  hipMalloc((void**)&temp, sizeof(float));
  hipMemcpy(temp,cuMetis_admin->tpwgts,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_tpwgts,nparts*sizeof(float));

  inittpwgts<<<nparts/32+1,32>>>(graph->cuda_tpwgts,temp,graph->cuda_nparts);

  hipFree(temp);

}



/*Malloc refine params*/
void cuMetis_malloc_refineinfo (cuMetis_admin_t *cuMetis_admin,\
cuMetis_graph_t *graph)
{
  int nvtxs=graph->nvtxs;
  int nparts=cuMetis_admin->nparts;

  hipMalloc((void**)&graph->cuda_bnd,nvtxs*sizeof(int));

  int num=0;

  hipMalloc((void**)&graph->cuda_bndnum,sizeof(int));
  hipMemcpy(graph->cuda_bndnum,&num,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_nparts,sizeof(int));
  hipMemcpy(graph->cuda_nparts,&nparts,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_pwgts,nparts*sizeof(int));

  initpwgts<<<nparts/32+1,32>>>(graph->cuda_pwgts,graph->cuda_nparts);
  
  Sumpwgts<<<nvtxs/32+1,32>>>(graph->cuda_pwgts,graph->cuda_where,graph->cuda_vwgt,graph->cuda_nvtxs);
  
  hipMalloc((void**)&graph->cuda_tvwgt,sizeof(int));
  hipMemcpy(graph->cuda_tvwgt,graph->tvwgt,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_tpwgts,nparts*sizeof(float));
  hipMalloc((void**)&graph->cuda_maxwgt,nparts*sizeof(int));
  hipMalloc((void**)&graph->cuda_minwgt,nparts*sizeof(int)); 

  float *temp;
  hipMalloc((void**)&temp, sizeof(float));
  hipMemcpy(temp,cuMetis_admin->tpwgts,sizeof(int),hipMemcpyHostToDevice);

  hipMalloc((void**)&graph->cuda_tpwgts,nparts*sizeof(float));
  inittpwgts<<<nparts/32+1,32>>>(graph->cuda_tpwgts,temp,graph->cuda_nparts);

  hipFree(temp);

}


/*CUDA-find vertex where ed-id>0 */
__global__ void Find_real_bnd_info(int *cuda_real_bnd_num, int *cuda_real_bnd, int *cuda_where, \
int *cuda_xadj, int *cuda_adjncy, int *cuda_adjwgt, int *cuda_nparts, int *nvtxs)
{
  int pi,me,other,i,me_part;
  pi=blockIdx.x*blockDim.x+threadIdx.x;

  if(pi<nvtxs[0]){
    me=0;
    other=0;
    me_part=cuda_where[pi];

    for(i=cuda_xadj[pi];i<cuda_xadj[pi+1];i++){
      if(cuda_where[cuda_adjncy[i]]==me_part){
        me+=cuda_adjwgt[i];
      }
      else{
        other+=cuda_adjwgt[i];
      }
    }
    if(other>me){
      cuda_real_bnd[atomicAdd(&cuda_real_bnd_num[0],1)]=pi;
    }
  }
}



/*CUDA-find boundary vertex should ro which part*/
__global__ void find_kayparams(int *cuda_real_bnd_num, int *bnd_info, int *cuda_real_bnd, int *cuda_where, \
int *cuda_xadj, int *cuda_adjncy, int *cuda_adjwgt, int *cuda_nparts, int *cuda_bn, int *cuda_bt, int *cuda_g)
{
  int ii,pi,other,i,me_wgt,other_wgt;
  int start,end;

  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<cuda_real_bnd_num[0]){
    pi=cuda_real_bnd[ii];
    start=(cuda_nparts[0])*ii;
    end=(cuda_nparts[0])*(ii+1);

    for(i=start;i<end;i++){
      bnd_info[i]=0;
    }

    for(i=cuda_xadj[pi];i<cuda_xadj[pi+1];i++){
      bnd_info[start+cuda_where[cuda_adjncy[i]]]+=cuda_adjwgt[i];
    }

    me_wgt=other_wgt=bnd_info[start+cuda_where[pi]];

    other=cuda_where[pi];

    for(i=start;i<end;i++){
      if(bnd_info[i]>other_wgt){
        other_wgt=bnd_info[i];
        other=i-start;
      }
    }

    cuda_g[ii]=other_wgt-me_wgt;
    cuda_bt[ii]=other;
    cuda_bn[ii]=pi;

  }
}



/*CUDA-init boundary vertex num*/
__global__ void initbndnum(int *n)
{
  n[0]=0;
}

int refine_pass=1;



/*CUDA-get a csr array*/
__global__ void findcsr(int *bt, int *n, int *nparts, int *bnd_num, int *a)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    n[2*ii]=-1;
    n[2*ii+1]=-1;

    for(int i=0;i<bnd_num[0];i++){
      if(ii==bt[i]){
        n[2*ii]=i;
        break; 
      }
    }

    if(n[2*ii]!=-1){
      for(int i=n[2*ii];i<bnd_num[0];i++){
        if(bt[i]!=ii){
          n[2*ii+1]=i-1;
          break; 
        }
      }
    }

    n[2*bt[bnd_num[0]-1]+1]=bnd_num[0]-1;

  }
}


/*CUDA-init params*/
__global__ void initcucsr(int *cu_csr,int *bndnum)
{
  cu_csr[0]=0;
  cu_csr[1]=bndnum[0];
}



/*Find boundary vertex information*/
void cuMetis_findgraphbndinfo(cuMetis_admin_t *cuMetis_admin,\
cuMetis_graph_t *graph)
{
  int nvtxs=graph->nvtxs;
  int nparts=cuMetis_admin->nparts;

  initbndnum<<<1,1>>>(graph->cuda_bndnum);

  Find_real_bnd_info<<<nvtxs/32+1,32>>>(graph->cuda_bndnum,graph->cuda_bnd,graph->cuda_where,\
    graph->cuda_xadj,graph->cuda_adjncy,graph->cuda_adjwgt,graph->cuda_nparts,graph->cuda_nvtxs); 
  
  int bnd_num; 
  hipMemcpy(&bnd_num,graph->cuda_bndnum, sizeof(int), hipMemcpyDeviceToHost);
  
  if(bnd_num>0){
    hipMalloc((void**)&graph->cuda_info, bnd_num*nparts* sizeof(int));
    
    find_kayparams<<<bnd_num/32+1,32>>>(graph->cuda_bndnum,graph->cuda_info,graph->cuda_bnd,graph->cuda_where,\
      graph->cuda_xadj,graph->cuda_adjncy,graph->cuda_adjwgt,graph->cuda_nparts,cu_bn,cu_bt,cu_g);
    
    initcucsr<<<1,1>>>(cu_csr,graph->cuda_bndnum);
    
    bb_segsort(cu_bt, cu_bn,bnd_num,cu_csr,  1);
    
    findcsr<<<nparts/32+1,32>>>(cu_bt,cu_que,graph->cuda_nparts,graph->cuda_bndnum,graph->cuda_nparts);
    
    hipFree(graph->cuda_info);
  }

  graph->cpu_bndnum=(int *)malloc(sizeof(int));
  graph->cpu_bndnum[0]=bnd_num;

}


/*CUDA-get the max/min pwgts*/
__global__ void Sum_maxmin_pwgts(int *cuda_maxwgt, int *cuda_minwgt, float *tpwgts, int *cuda_tvwgt, int *a)
{
  int ii;
  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    float ubfactor=1.03;

    cuda_maxwgt[ii]=int(tpwgts[ii]*cuda_tvwgt[0]*ubfactor);
    cuda_minwgt[ii]=int(tpwgts[ii]*cuda_tvwgt[0]/ ubfactor);
  }
}



/*CUDA-move vertex*/
__global__ void Exnode_part1(int *que, int *pwgts, int *bndnum, int *bnd, int *bndto, int *vwgt,\
  int *maxvwgt, int *minvwgt, int *where, int *a)
{
  int me,to,vvwgt,memax,memin,tomax,tomin;
  int nmoves=0;
  int i,ii;

  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    if(que[2*ii]!=-1){
      for(i=que[2*ii];i<=que[2*ii+1];i++){
        vvwgt=vwgt[bnd[i]];
        me=where[bnd[i]];
        to=bndto[i];

        memax=maxvwgt[me];
        memin=minvwgt[me];
        tomax=maxvwgt[to];
        tomin=minvwgt[to];

        if(me<=to){
          if(((pwgts[to]+vvwgt>=tomin)&&(pwgts[to]+vvwgt<=tomax))\
          &&((pwgts[me]-vvwgt>=memin)&&(pwgts[me]-vvwgt<=memax))){
            atomicAdd(&pwgts[to],vvwgt);
            atomicSub(&pwgts[me],vvwgt);
            where[bnd[i]]=to;
            nmoves++;
          }
        }
      }
    }
  }
}



/*CUDA-move vertex*/
__global__ void Exnode_part2(int *que, int *pwgts, int *bndnum, int *bnd, int *bndto, int *vwgt,\
  int *maxvwgt, int *minvwgt, int *where, int *a)
{
  int me,to,vvwgt,memax,memin,tomax,tomin;
  int nmoves=0;
  int i,ii;

  ii=blockIdx.x*blockDim.x+threadIdx.x;

  if(ii<a[0]){
    if(que[2*ii]!=-1){
      for(i=que[2*ii];i<=que[2*ii+1];i++){
        vvwgt=vwgt[bnd[i]];
        me=where[bnd[i]];
        to=bndto[i];

        memax=maxvwgt[me];
        memin=minvwgt[me];
        tomax=maxvwgt[to];
        tomin=minvwgt[to];

        if(me>to){
          if(((pwgts[to]+vvwgt>=tomin)&&(pwgts[to]+vvwgt<=tomax))\
          &&((pwgts[me]-vvwgt>=memin)&&(pwgts[me]-vvwgt<=memax))){    
            atomicAdd(&pwgts[to],vvwgt);
            atomicSub(&pwgts[me],vvwgt);
            where[bnd[i]]=to;
            nmoves++;
          }
        }
      }
    }
  }
}




/*Graph multilevel uncoarsening algorithm*/
void cuMetis_uncoarsen(cuMetis_admin_t *cuMetis_admin, cuMetis_graph_t *graph)
{
  int nparts=cuMetis_admin->nparts;

  Sum_maxmin_pwgts<<<nparts/32+1,32>>>(graph->cuda_maxwgt,graph->cuda_minwgt,\
  graph->cuda_tpwgts,graph->cuda_tvwgt,graph->cuda_nparts);

  for(int i=0;i<refine_pass;i++){
    cuMetis_findgraphbndinfo(cuMetis_admin,graph);

    if(graph->cpu_bndnum[0]>0){
      Exnode_part1<<<nparts/32+1,32>>>(cu_que,graph->cuda_pwgts,graph->cuda_bndnum,cu_bn,cu_bt,graph->cuda_vwgt,\
        graph->cuda_maxwgt,graph->cuda_minwgt,graph->cuda_where,graph->cuda_nparts);  
      
      Exnode_part2<<<nparts/32+1,32>>>(cu_que,graph->cuda_pwgts,graph->cuda_bndnum,cu_bn,cu_bt,graph->cuda_vwgt,\
        graph->cuda_maxwgt,graph->cuda_minwgt,graph->cuda_where,graph->cuda_nparts);   
    }
    else
    break;

  }
}



/*Free graph uncoarsening phase params*/
void cuMetis_free_uncoarsen(cuMetis_graph_t *graph)
{
  hipFree(graph->cuda_xadj);
  hipFree(graph->cuda_cmap);
  hipFree(graph->cuda_nvtxs);
  hipFree(graph->cuda_adjncy);
  hipFree(graph->cuda_adjwgt);
  hipFree(graph->cuda_vwgt);
  hipFree(graph->cuda_maxwgt);
  hipFree(graph->cuda_minwgt);
  hipFree(graph->cuda_where);
  hipFree(graph->cuda_pwgts);
  hipFree(graph->cuda_bnd);
  hipFree(graph->cuda_bndnum);
  hipFree(graph->cuda_real_bnd_num);
  hipFree(graph->cuda_real_bnd);
  hipFree(graph->cuda_nparts);
  hipFree(graph->cuda_tvwgt);
  hipFree(graph->cuda_tpwgts);
}



/*Graph kway-partition algorithm*/
int cuMetis_kway_partition(cuMetis_admin_t *cuMetis_admin, \
cuMetis_graph_t *graph, int *part)
{
  cuMetis_graph_t *cgraph;

  hipDeviceSynchronize();
  gettimeofday(&begin_part_coarsen,NULL);
  cgraph = cuMetis_coarsen(cuMetis_admin, graph);
  hipDeviceSynchronize();
  gettimeofday(&end_part_coarsen,NULL);
  part_coarsen += (end_part_coarsen.tv_sec - begin_part_coarsen.tv_sec) * 1000 + (end_part_coarsen.tv_usec - begin_part_coarsen.tv_usec) / 1000.0;
  
  // printf("cnvtxs=%d\n",cgraph->nvtxs);

  hipDeviceSynchronize();
  gettimeofday(&begin_part_init,NULL);
  cuMetis_initialpartition(cuMetis_admin, cgraph);
  hipDeviceSynchronize();
  gettimeofday(&end_part_init,NULL);
  part_init += (end_part_init.tv_sec - begin_part_init.tv_sec) * 1000 + (end_part_init.tv_usec - begin_part_init.tv_usec) / 1000.0;

  hipDeviceSynchronize();
  gettimeofday(&begin_part_uncoarsen,NULL);
  Mallocinit_refineinfo(cuMetis_admin,cgraph);

  hipDeviceSynchronize();
  gettimeofday(&begin_part_krefine,NULL);

  cuMetis_uncoarsen(cuMetis_admin,cgraph);

  hipDeviceSynchronize();
  gettimeofday(&end_part_krefine,NULL);
  part_krefine += (end_part_krefine.tv_sec - begin_part_krefine.tv_sec) * 1000 + (end_part_krefine.tv_usec - begin_part_krefine.tv_usec) / 1000.0;
  

  for(int i=0;;i++){
    if(cgraph!=graph){
      cgraph=cgraph->finer;

      hipMalloc((void**)&cgraph->cuda_where, cgraph->nvtxs*sizeof(int));

      hipDeviceSynchronize();
      gettimeofday(&begin_part_map,NULL);

      cuMetis_kway_project(cuMetis_admin,cgraph);

      hipDeviceSynchronize();
      gettimeofday(&end_part_map,NULL);
      part_map += (end_part_map.tv_sec - begin_part_map.tv_sec) * 1000 + (end_part_map.tv_usec - begin_part_map.tv_usec) / 1000.0;

      cuMetis_malloc_refineinfo(cuMetis_admin,cgraph);   

      hipDeviceSynchronize();
      gettimeofday(&begin_part_krefine,NULL);

      cuMetis_uncoarsen(cuMetis_admin,cgraph);

      hipDeviceSynchronize();
      gettimeofday(&end_part_krefine,NULL);
      part_krefine += (end_part_krefine.tv_sec - begin_part_krefine.tv_sec) * 1000 + (end_part_krefine.tv_usec - begin_part_krefine.tv_usec) / 1000.0;

      cuMetis_free_uncoarsen(cgraph->coarser);
    } 
    else 
      break; 
  }

  hipDeviceSynchronize();
  gettimeofday(&end_part_uncoarsen,NULL);
  part_uncoarsen += (end_part_uncoarsen.tv_sec - begin_part_uncoarsen.tv_sec) * 1000 + (end_part_uncoarsen.tv_usec - begin_part_uncoarsen.tv_usec) / 1000.0;
  
  return 0;
}




/*Graph partition algorithm*/
int cuMetis_PartGraph(int *nvtxs,  int *xadj, int *adjncy, int *vwgt,int *adjwgt, \
int *nparts, float *tpwgts, float *ubvec, int *objval, int *part)
{
  cuMetis_graph_t *graph;
  cuMetis_admin_t *cuMetis_admin;

  cuMetis_admin = cuMetis_set_graph_admin( *nparts, tpwgts, ubvec);

  graph = cuMetis_set_graph(cuMetis_admin, *nvtxs, xadj, adjncy, vwgt, adjwgt);

  cuMetis_set_kway_bal(cuMetis_admin, graph);

  cuMetis_admin->Coarsen_threshold = cuMetis_max((*nvtxs)/(20*cuMetis_compute_log2(*nparts)),30*(*nparts));
  
  cuMetis_admin->nIparts=(cuMetis_admin->Coarsen_threshold==30*(*nparts) ?4 :5);
  
  test_time=1;
  
  cuMetis_malloc_original_coarseninfo(cuMetis_admin,graph);  
  
  hipMalloc((void**)&cu_bn, graph->nvtxs* sizeof(int));
  hipMalloc((void**)&cu_bt, graph->nvtxs* sizeof(int));
  hipMalloc((void**)&cu_g, graph->nvtxs*sizeof(int));
  hipMalloc((void**)&cu_csr, 2*sizeof(int));
  hipMalloc((void**)&cu_que, 2*cuMetis_admin->nparts*sizeof(int));   
  
  hipDeviceSynchronize();
  gettimeofday(&begin_part_all,NULL);
  for(int i=0;i<test_time;i++){
    *objval=cuMetis_kway_partition(cuMetis_admin,graph,part);
  }
  hipDeviceSynchronize();
  gettimeofday(&end_part_all,NULL);
  part_all += (end_part_all.tv_sec - begin_part_all.tv_sec) * 1000 + (end_part_all.tv_usec - begin_part_all.tv_usec) / 1000.0;
  
  hipMemcpy(part,graph->cuda_where, graph->nvtxs*sizeof(int), hipMemcpyDeviceToHost);

  cuMetis_free_coarsen(graph);
  cuMetis_free_uncoarsen(graph);

  hipFree(cu_bn);
  hipFree(cu_bt);
  hipFree(cu_g);
  hipFree(cu_csr);
  hipFree(cu_que);

  return 1;

}



/*Error exit*/
void cuMetis_error_exit(char *f_str,...)
{
  va_list a;
  va_start(a,f_str);
  vfprintf(stderr,f_str,a);
  va_end(a);

  if (strlen(f_str)==0||f_str[strlen(f_str)-1]!='\n'){
    fprintf(stderr,"\n");
  }

  fflush(stderr);

  if(1)
    exit(-2);
}



/*Open file*/
FILE *cuMetis_fopen(char *fname, char *mode, const char *msg)
{
  FILE *fp;
  char error_message[8192];
  fp=fopen(fname, mode);
  if(fp!=NULL){
    return fp;
  }
  sprintf(error_message,"file: %s, mode: %s, [%s]",fname,mode,msg);
  perror(error_message);
  cuMetis_error_exit("Failed on file fopen()\n");
  return NULL;
}



/*Read graph file*/
cuMetis_graph_t *cuMetis_readgraph(char *filename)
{
  int i,k,fmt,nfields,readew,readvw,readvs,edge,ewgt;
  int *xadj,*adjncy,*vwgt,*adjwgt;
  char *line=NULL,fmtstr[256],*curstr,*newstr;
  size_t lnlen=0;
  FILE *fpin;

  cuMetis_graph_t *graph;
  graph = cuMetis_create_cpu_graph();

  fpin = cuMetis_fopen(filename,"r","Readgraph: Graph");

  do{
    if(getline(&line,&lnlen,fpin)==-1){ 
      cuMetis_error_exit("Premature end of input file: file: %s\n", filename);
    }
  }while(line[0]=='%');

  fmt= 0;
  nfields = sscanf(line, "%d %d %d", &(graph->nvtxs), &(graph->nedges), &fmt);

  if(nfields<2){
    cuMetis_error_exit("The input file does not specify the number of vertices and edges.\n");
  }

  if(graph->nvtxs<=0||graph->nedges<=0){
   cuMetis_error_exit("The supplied nvtxs:%d and nedges:%d must be positive.\n",graph->nvtxs,graph->nedges);
  }

  if(fmt>111){ 
    cuMetis_error_exit("Cannot read this type of file format [fmt=%d]!\n",fmt);
  }

  sprintf(fmtstr,"%03d",fmt%1000);
  readvs=(fmtstr[0]=='1');
  readvw=(fmtstr[1]=='1');
  readew=(fmtstr[2]=='1');

  graph->nedges *=2;

  xadj=graph->xadj=(int*)malloc(sizeof(int)*(graph->nvtxs+1));
  for(i=0;i<graph->nvtxs+1;i++){
    xadj[i]=graph->xadj[i]=0;
  }

  adjncy=graph->adjncy=(int*)malloc(sizeof(int)*(graph->nedges));

  vwgt=graph->vwgt= (int*)malloc(sizeof(int)*(graph->nvtxs));

  for(i=0;i<graph->nvtxs;i++){
    vwgt[i]=graph->vwgt[i]=1;
  }

  adjwgt = graph->adjwgt=(int*)malloc(sizeof(int)*(graph->nedges));
  for(i=0;i<graph->nedges;i++){
    adjwgt[i]=graph->adjwgt[i]=1;
  }

  for(xadj[0]=0,k=0,i=0;i<graph->nvtxs;i++){
    do{
      if(getline(&line,&lnlen,fpin)==-1){
      cuMetis_error_exit("Premature end of input file while reading vertex %d.\n", i+1);
      } 
    }while(line[0]=='%');

    curstr=line;
    newstr=NULL;

    if(readvw){
      vwgt[i]=strtol(curstr, &newstr, 10);

      if(newstr==curstr){
        cuMetis_error_exit("The line for vertex %d does not have enough weights "
          "for the %d constraints.\n", i+1, 1);
      }
      if(vwgt[i]<0){
        cuMetis_error_exit("The weight vertex %d and constraint %d must be >= 0\n", i+1, 0);
      }
      curstr = newstr;
    }

    while(1){
      edge=strtol(curstr,&newstr,10);
      if(newstr==curstr){
        break; 
      }

      curstr=newstr;
      if (edge< 1||edge>graph->nvtxs){
        cuMetis_error_exit("Edge %d for vertex %d is out of bounds\n",edge,i+1);
      }

      ewgt=1;

      if(readew){
        ewgt=strtol(curstr,&newstr,10);

        if(newstr==curstr){
          cuMetis_error_exit("Premature end of line for vertex %d\n", i+1);
        }

        if(ewgt<=0){
          cuMetis_error_exit("The weight (%d) for edge (%d, %d) must be positive.\n",    ewgt, i+1, edge);
        }

        curstr=newstr;
      }

      if(k==graph->nedges){
        cuMetis_error_exit("There are more edges in the file than the %d specified.\n", graph->nedges/2);
      }

      adjncy[k]=edge-1;
      adjwgt[k]=ewgt;
      k++;

    } 
    xadj[i+1]=k;

  }
  fclose(fpin);

  if(k!=graph->nedges){
    printf("------------------------------------------------------------------------------\n");
    printf("***  I detected an error in your input file  ***\n\n");
    printf("In the first line of the file, you specified that the graph contained\n"
      "%d edges. However, I only found %d edges in the file.\n", graph->nedges/2,k/2);
    if(2*k==graph->nedges){
      printf("\n *> I detected that you specified twice the number of edges that you have in\n");
      printf("    the file. Remember that the number of edges specified in the first line\n");
      printf("    counts each edge between vertices v and u only once.\n\n");
    }
    printf("Please specify the correct number of edges in the first line of the file.\n");
    printf("------------------------------------------------------------------------------\n");
    exit(0);
  }
  free(line);
  return graph;
}



/*Write to file*/
void cuMetis_writetofile(char *fname, int *part, int n, int nparts)
{
  FILE *fpout;
  int i;
  char filename[1280000];
  sprintf(filename, "%s.part.%d", fname, nparts);

  fpout = cuMetis_fopen(filename, "w", __func__);

  for (i=0; i<n; i++){
    fprintf(fpout,"%d\n",part[i]);
  }

  fclose(fpout);
}


/*Main function*/
int main(int argc, char **argv)
{  
  hipSetDevice(1);

  int i;
  int nparts;
  char *filename=(argv[1]);
  nparts =atoi(argv[2]);

  cuMetis_graph_t *graph;

  int *part;
  int objval;
  graph=cuMetis_readgraph(filename); 

  int c;
  c=nparts;
  float tpwgts[c];
  for(i=0;i<c;i++){
    tpwgts[i]=1/c;
  }
  
  float ubvec=1.03;
  part=(int*)malloc(sizeof(int)*(graph->nvtxs));
  
  cuMetis_PartGraph(&graph->nvtxs, graph->xadj, graph->adjncy, graph->vwgt,graph->adjwgt, &nparts, tpwgts, &ubvec,  &objval, part);
  
  printf("Graph: %s \nVertex: %d Edge: %d\n",filename,graph->nvtxs,graph->nedges);

  printf("cuMetis-Partition-end\n");
  printf("cuMetis_Partition_time= %lf     ms\n",part_all);
  printf("------Coarsen_time=          %lf     ms\n",part_coarsen);
  printf("------Init_time=             %lf     ms\n",part_init);
  printf("------Uncoarsen_time=        %lf     ms\n",part_uncoarsen);
  
  //cuMetis_writetofile(filename, part, graph->nvtxs, nparts); 

  printf("------------------------------------------\n");
  printf("The match pattern=           %lf     ms\n",part_match+part_cmatch);
  printf("The multi-node pattern=      %lf     ms\n",part_contract+part_ccontract+part_2map+part_map);
  printf("The 2refine pattern=         %lf     ms\n",part_2refine);
  printf("The krefine pattern=         %lf     ms\n",part_krefine);
  printf("Bfs+Slipt=                   %lf     ms\n",part_bfs+part_slipt);

  int e=cuMetis_computecut(graph, part);
  printf("Edge-cut=                    %d\n",e);
}

